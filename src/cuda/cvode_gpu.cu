#include "hip/hip_runtime.h"
/* Copyright (C) 2021 Barcelona Supercomputing Center and University of
 * Illinois at Urbana-Champaign
 * SPDX-License-Identifier: MIT
 */
#include "cvode_cuda.h"

//#define PROFILE_GPU_SOLVING

extern "C" {
#include "cvode_gpu.h"
}
#ifdef TRACE_CPUGPU
#include "nvToolsExt.h"
#endif

#ifdef CAMP_USE_MPI
#include <mpi.h>
#endif

int cudaCVode(void *cvode_mem, double t_final, N_Vector yout,
               SolverData *sd, double t_initial){
  ModelDataGPU *mGPU = sd->mGPU;
  ModelData *md = &(sd->model_data);
  hipStream_t stream;
  hipStreamCreate(&stream);
#ifdef PROFILE_GPU_SOLVING
  hipEventRecord(sd->startGPU,stream);
#endif
  int n_cells=md->n_cells_gpu;
  hipMemcpyAsync(mGPU->rxn_env_data,md->rxn_env_data,md->n_rxn_env_data * n_cells * sizeof(double),hipMemcpyHostToDevice,stream);
  hipMemcpyAsync(mGPU->state,md->total_state,md->n_per_cell_state_var*n_cells*sizeof(double),hipMemcpyHostToDevice,stream);
  mGPU->init_time_step = sd->init_time_step;
  mGPU->tout = t_final;
  cvodeRun(t_initial, mGPU, n_cells, md->n_per_cell_dep_var, stream); //Asynchronous
  hipMemcpyAsync(md->total_state, mGPU->state, md->n_per_cell_state_var*md->n_cells_gpu * sizeof(double), hipMemcpyDeviceToHost, stream);
#ifdef PROFILE_GPU_SOLVING
  hipEventRecord(sd->stopGPU,stream);
#endif
  //CPU
#ifdef TRACE_CPUGPU
  nvtxRangePushA("CPU Code");
#endif
#ifdef PROFILE_GPU_SOLVING
  double startTime = MPI_Wtime();
#endif
  n_cells=md->n_cells;
  int flag=CV_SUCCESS;
  int n_state_var = md->n_per_cell_state_var;
  double *state = md->total_state;
  double *env = md->total_env;
  double *rxn_env_data = md->rxn_env_data;
  md->total_state += n_state_var*md->n_cells_gpu;
  md->total_env += CAMP_NUM_ENV_PARAM_*md->n_cells_gpu;
  md->rxn_env_data += md->n_rxn_env_data*md->n_cells_gpu;
  for (int i_cell = md->n_cells_gpu; i_cell < n_cells; i_cell++) {
    int i_dep_var = 0;
    for (int i_spec = 0; i_spec < n_state_var; i_spec++) {
      if (sd->model_data.var_type[i_spec] == CHEM_SPEC_VARIABLE) {
        NV_Ith_S(sd->y, i_dep_var++) =
            md->total_state[i_spec] > TINY
           ? (realtype)md->total_state[i_spec] : TINY;
      }
    }
    if (sd->is_reset_jac == 1) {
      N_VConst(0.0, md->J_state);
      N_VConst(0.0, md->J_deriv);
      SM_NNZ_S(md->J_solver) = SM_NNZ_S(md->J_init);
      for (int i = 0; i < SM_NNZ_S(md->J_solver); i++) {
        (SM_DATA_S(md->J_solver))[i] = 0.0;
      }
    }
    flag = CVodeReInit(sd->cvode_mem, t_initial, sd->y);
    flag = SUNKLUReInit(sd->ls, sd->J, SM_NNZ_S(sd->J), SUNKLU_REINIT_PARTIAL);
    flag = CVodeSetInitStep(sd->cvode_mem, sd->init_time_step);
    realtype t_rt = (realtype)t_initial;
    flag=0;
    flag = CVode(sd->cvode_mem, t_final, sd->y, &t_rt, CV_NORMAL);
    if (flag < 0){
      flag = CAMP_SOLVER_FAIL;
      break;
    }
    i_dep_var = 0;
    for (int i_spec = 0; i_spec < n_state_var; i_spec++) {
      if (md->var_type[i_spec] == CHEM_SPEC_VARIABLE) {
        md->total_state[i_spec] =
          (double)(NV_Ith_S(sd->y, i_dep_var) > 0.0
           ? NV_Ith_S(sd->y, i_dep_var)
           : 0.0);
        i_dep_var++;
      }
    }
    md->total_state += n_state_var;
    md->total_env += CAMP_NUM_ENV_PARAM_;
    md->rxn_env_data += md->n_rxn_env_data;
  }
  md->total_state  = state;
  md->total_env = env;
  md->rxn_env_data = rxn_env_data;
#ifdef PROFILE_GPU_SOLVING
  double timeCPU = (MPI_Wtime() - startTime);
#endif
#ifdef TRACE_CPUGPU
  nvtxRangePop();
#endif
#ifdef PROFILE_GPU_SOLVING
  hipEventRecord(sd->startGPUSync,stream);
#endif
  hipStreamSynchronize(stream);
  hipDeviceSynchronize();
#ifdef PROFILE_GPU_SOLVING
  hipEventRecord(sd->stopGPUSync,stream);
  hipEventSynchronize(sd->stopGPUSync);
  hipEventSynchronize(sd->stopGPU);
  float msDevice = 0.0;
  hipEventElapsedTime(&msDevice, sd->startGPU, sd->stopGPU);
  double timeGPU=msDevice/1000;
  hipEventElapsedTime(&msDevice, sd->startGPUSync, sd->stopGPUSync);
  timeGPU+=msDevice/1000;
  MPI_Barrier(MPI_COMM_WORLD);
  int rank;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  double concurrency=100;
  double min=fmin(timeGPU,timeCPU);
  double max=fmax(timeGPU,timeCPU);
  if(min!=max) concurrency=min/max;
  if(rank==0)printf("Concurrency: %.2lf%% weight_gpu: %.2lf%%\n",concurrency,sd->weight_gpu);
  //for example: [ W ] weight_gpu = 90% -> [ C ] concurrency = 50%, prev_w=100%, prev_c=0% -> df = -10%
  //W = 80% , C = 20% , df = -20% -> best_c= 100%, df = df * (best_c/concurrency) = -10+10*(100/50)=-20%, weight_gpu = 100+df = 80%
  //Another example: [ W ] weight_gpu = 90% -> [ C ] concurrency = 60%,
  //W = 80% , C = 20% , df = -10% -> best_c= 100%, df = df * (best_c/concurrency) = -10*(100/60)=-16.67%, weight_gpu = 100+df = 83.33%
  //first df : sd->df = sd->weight_gpu-100;
  //sd->df = (int)sd->df * (100/concurrency);
  //printf("next sd->df %d 100/conc %lf\n",sd->df, 100/concurrency);
  //sd->weight_gpu = 100+sd->df;
  //md->n_cells_gpu = (int)(n_cells*sd->weight_gpu/100);

  //The plan is to use a delta_factor that sums to the current weight_gpu, and multiply by a perc this delta factor
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  printf("DEBUG: CAMP_PROFILE_DEVICE_FUNCTIONS\n");
  hipMemcpyAsync(&mCPU->mdvCPU, mGPU->mdvo, sizeof(ModelDataVariable), hipMemcpyDeviceToHost, stream);
#endif
#endif
  hipStreamDestroy(stream);
  return(CV_SUCCESS);
}

void solver_get_statistics_gpu(SolverData *sd){
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  ModelDataGPU *mGPU = sd->mGPU;
  ModelDataCPU *mCPU = &(sd->mCPU);
  hipMemcpy(&mCPU->mdvCPU,mGPU->mdvo,sizeof(ModelDataVariable),hipMemcpyDeviceToHost);
#endif
}