/* Copyright (C) 2021 Barcelona Supercomputing Center and University of
 * Illinois at Urbana-Champaign
 * SPDX-License-Identifier: MIT
 */
#include "cvode_cuda.h"

extern "C" {
#include "cvode_gpu.h"
}
#ifdef TRACE_CPUGPU
#include "nvToolsExt.h"
#endif

#ifdef CAMP_USE_MPI
#include <mpi.h>
#endif

int cudaCVode(void *cvode_mem, double t_final, N_Vector yout,
               SolverData *sd, double t_initial){
  CVodeMem cv_mem = (CVodeMem) cvode_mem;
  ModelDataGPU *mGPU = sd->mGPU;
  ModelData *md = &(sd->model_data);
#ifdef CAMP_PROFILE_SOLVING
  hipEventRecord(sd->startcvStep);
#endif
  hipStream_t stream;
  hipStreamCreate(&stream);
  int n_cells=md->n_cells_gpu;
  hipMemcpyAsync(mGPU->rxn_env_data,md->rxn_env_data,md->n_rxn_env_data * n_cells * sizeof(double),hipMemcpyHostToDevice,stream);
  hipMemcpyAsync(mGPU->state,md->total_state,md->n_per_cell_state_var*n_cells*sizeof(double),hipMemcpyHostToDevice,stream);
  mGPU->init_time_step = sd->init_time_step;
  mGPU->tout = t_final;
  cvodeRun(t_initial, mGPU, n_cells, md->n_per_cell_dep_var, stream); //Asynchronous
  //CPU
#ifdef TRACE_CPUGPU
  nvtxRangePushA("CPU Code");
#endif
  n_cells=md->n_cells;
  int flag=CV_SUCCESS;
  int n_state_var = md->n_per_cell_state_var;
  double *state = md->total_state;
  double *env = md->total_env;
  double *rxn_env_data = md->rxn_env_data;
  md->total_state += n_state_var*md->n_cells_gpu;
  md->total_env += CAMP_NUM_ENV_PARAM_*md->n_cells_gpu;
  md->rxn_env_data += md->n_rxn_env_data*md->n_cells_gpu;
  for (int i_cell = md->n_cells_gpu; i_cell < n_cells; i_cell++) {
    int i_dep_var = 0;
    for (int i_spec = 0; i_spec < n_state_var; i_spec++) {
      if (sd->model_data.var_type[i_spec] == CHEM_SPEC_VARIABLE) {
        NV_Ith_S(sd->y, i_dep_var++) =
            md->total_state[i_spec] > TINY
           ? (realtype)md->total_state[i_spec] : TINY;
      }
    }
    if (sd->is_reset_jac == 1) {
      N_VConst(0.0, md->J_state);
      N_VConst(0.0, md->J_deriv);
      SM_NNZ_S(md->J_solver) = SM_NNZ_S(md->J_init);
      for (int i = 0; i < SM_NNZ_S(md->J_solver); i++) {
        (SM_DATA_S(md->J_solver))[i] = 0.0;
      }
    }
    flag = CVodeReInit(sd->cvode_mem, t_initial, sd->y);
    flag = SUNKLUReInit(sd->ls, sd->J, SM_NNZ_S(sd->J), SUNKLU_REINIT_PARTIAL);
    flag = CVodeSetInitStep(sd->cvode_mem, sd->init_time_step);
    realtype t_rt = (realtype)t_initial;
    flag=0;
    flag = CVode(sd->cvode_mem, t_final, sd->y, &t_rt, CV_NORMAL);
    if (flag < 0){
      flag = CAMP_SOLVER_FAIL;
      break;
    }
    i_dep_var = 0;
    for (int i_spec = 0; i_spec < n_state_var; i_spec++) {
      if (md->var_type[i_spec] == CHEM_SPEC_VARIABLE) {
        md->total_state[i_spec] =
          (double)(NV_Ith_S(sd->y, i_dep_var) > 0.0
           ? NV_Ith_S(sd->y, i_dep_var)
           : 0.0);
        i_dep_var++;
      }
    }
    md->total_state += n_state_var;
    md->total_env += CAMP_NUM_ENV_PARAM_;
    md->rxn_env_data += md->n_rxn_env_data;
  }
  md->total_state  = state;
  md->total_env = env;
  md->rxn_env_data = rxn_env_data;
  hipMemcpyAsync(md->total_state, mGPU->state, md->n_per_cell_state_var*md->n_cells_gpu * sizeof(double), hipMemcpyDeviceToHost, stream);
#ifdef TRACE_CPUGPU
  nvtxRangePop();
#endif
  hipDeviceSynchronize();
#ifdef CAMP_PROFILE_SOLVING
    hipEventRecord(sd->stopcvStep);
    hipEventSynchronize(sd->stopcvStep);
    float mscvStep = 0.0;
    hipEventElapsedTime(&mscvStep, sd->startcvStep, sd->stopcvStep);
    cv_mem->timecvStep+= mscvStep/1000;
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    hipMemcpy(&mCPU->mdvCPU, mGPU->mdvo, sizeof(ModelDataVariable), hipMemcpyDeviceToHost);
#endif
#endif
  hipStreamDestroy(stream);
  return(CV_SUCCESS);
}

void solver_get_statistics_gpu(SolverData *sd){
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  ModelDataGPU *mGPU = sd->mGPU;
  ModelDataCPU *mCPU = &(sd->mCPU);
  hipMemcpy(&mCPU->mdvCPU,mGPU->mdvo,sizeof(ModelDataVariable),hipMemcpyDeviceToHost);
#endif
}