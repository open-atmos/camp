#include "hip/hip_runtime.h"
/* Copyright (C) 2021 Barcelona Supercomputing Center and University of
* Illinois at Urbana-Champaign
* SPDX-License-Identifier: MIT
*/

#include "cvode_cuda.h"

__device__
void print_double(double *x, int len, const char *s){
#ifdef USE_PRINT_ARRAYS
  __syncthreads();
  if(threadIdx.x==0 && blockIdx.x==0){
    for (int i=0; i<len; i++){
      printf("%s[%d]=%.17le\n",s,i,x[i]);
    }
  }
  __syncthreads();
#endif
}

__device__
void print_int(int *x, int len, const char *s){
#ifdef USE_PRINT_ARRAYS
  __syncthreads();
  if(threadIdx.x==0 && blockIdx.x==0){
    for (int i=0; i<len; i++){
      printf("%s[%d]=%d\n",s,i,x[i]);
    }
  }
  __syncthreads();
#endif
}

__device__
double dSUNRpowerR(double base, double exponent){
  if (base <= ZERO) return(ZERO);
#ifdef EQUALLIZE_CPU_CUDA_POW
  if(exponent==(1./2)) return sqrt(base);
  if(exponent==(1./3)) return sqrt(sqrt(base));
  if(exponent==(1./4)) return sqrt(sqrt(base));
#endif
  return(pow(base, (double)(exponent)));
}

__device__
double dSUNRpowerI(double base, int exponent)
{
  int i, expt;
  double prod;
  prod = ONE;
  expt = abs(exponent);
  for(i = 1; i <= expt; i++) prod *= base;
  if (exponent < 0) prod = ONE/prod;
  return(prod);
}

#ifdef IS_DEBUG_MODE_removeAtomic

__device__
void time_derivative_add_value_gpu(TimeDerivativeGPU time_deriv, unsigned int spec_id,
                               double rate_contribution) {
  if (rate_contribution > 0.0) {
    time_deriv.production_rates[spec_id] += rate_contribution;
  } else {
    time_deriv.loss_rates[spec_id] += -rate_contribution;
  }
}

#else

__device__
void time_derivative_add_value_gpu(TimeDerivativeGPU time_deriv, unsigned int spec_id,
                               double rate_contribution) {
  if (rate_contribution > 0.0) {
    atomicAdd_block(&(time_deriv.production_rates[spec_id]),rate_contribution);
  } else {
    atomicAdd_block(&(time_deriv.loss_rates[spec_id]),-rate_contribution);
  }
}

#endif

__device__
void rxn_gpu_first_order_loss_calc_deriv_contrib(ModelDataVariable *sc, TimeDerivativeGPU time_deriv, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double rate = rxn_env_data[0] * sc->grid_cell_state[int_data[1]-1];
  if (int_data[2] >= 0) time_derivative_add_value_gpu(time_deriv, int_data[2], -rate);
}

__device__
void rxn_gpu_CMAQ_H2O2_calc_deriv_contrib(ModelDataVariable *sc, TimeDerivativeGPU time_deriv, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double rate = rxn_env_data[0];
  for (int i_spec=0; i_spec<int_data[0]; i_spec++) rate *= sc->grid_cell_state[int_data[(2 + i_spec)]-1];
  if (rate!=0.) {
    int i_dep_var = 0;
    for (int i_spec=0; i_spec<int_data[0]; i_spec++, i_dep_var++) {
      if (int_data[(2 + int_data[0]+int_data[1]+i_dep_var)] < 0) continue;
      time_derivative_add_value_gpu(time_deriv, int_data[(2 + int_data[0]+int_data[1]+i_dep_var)], -rate);
    }
    for (int i_spec=0; i_spec<int_data[1]; i_spec++, i_dep_var++) {
      if (int_data[(2 + int_data[0]+int_data[1]+i_dep_var)] < 0) continue;
      if (-rate*float_data[(7 + i_spec)]*time_step <= sc->grid_cell_state[int_data[(2 + int_data[0]+ i_spec)]-1]) {
        time_derivative_add_value_gpu(time_deriv, int_data[(2 + int_data[0]+int_data[1]+i_dep_var)],rate*float_data[(7 + i_spec)]);
      }
    }
  }
}

__device__
void rxn_gpu_CMAQ_OH_HNO3_calc_deriv_contrib(ModelDataVariable *sc, TimeDerivativeGPU time_deriv, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double rate = rxn_env_data[0];
  for (int i_spec=0; i_spec<int_data[0]; i_spec++) rate *= sc->grid_cell_state[int_data[(2 + i_spec)]-1];
  if (rate!=0.) {
    int i_dep_var = 0;
    for (int i_spec=0; i_spec<int_data[0]; i_spec++, i_dep_var++) {
      if (int_data[(2 + int_data[0] + int_data[1] + i_dep_var)] < 0) continue;
      time_derivative_add_value_gpu(time_deriv, int_data[(2 + int_data[0] + int_data[1] + i_dep_var)], -rate);
    }
    for (int i_spec=0; i_spec<int_data[1]; i_spec++, i_dep_var++) {
      if (int_data[(2 + int_data[0] + int_data[1] + i_dep_var)] < 0) continue;
      if (-rate*float_data[(11 + i_spec)]*time_step <= sc->grid_cell_state[int_data[(2 + int_data[0] + i_spec)]-1]) {
        time_derivative_add_value_gpu(time_deriv, int_data[(2 + int_data[0] + int_data[1] + i_dep_var)],rate*float_data[(11 + i_spec)]);
      }
    }
  }
}

__device__
void rxn_gpu_arrhenius_calc_deriv_contrib(ModelDataVariable *sc, TimeDerivativeGPU time_deriv,
                                int *rxn_int_data, double *rxn_float_data,
                                double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double rate = rxn_env_data[0];
  for (int i_spec=0; i_spec<int_data[0]; i_spec++)
    rate *= sc->grid_cell_state[int_data[(2 + i_spec)]-1];
  if (rate!=0.) {
    int i_dep_var = 0;
    for (int i_spec=0; i_spec<int_data[0]; i_spec++, i_dep_var++) {
      if (int_data[2 + int_data[0] + int_data[1] + i_dep_var] < 0) continue;
      time_derivative_add_value_gpu(time_deriv, int_data[2 + int_data[0] + int_data[1] + i_dep_var], -rate);
    }
    for (int i_spec=0; i_spec<int_data[1]; i_spec++, i_dep_var++) {
      if (int_data[2 + int_data[0] + int_data[1] + i_dep_var] < 0) continue;
      if (-rate*float_data[6+i_spec]*time_step <= sc->grid_cell_state[int_data[(2 + int_data[0] + i_spec)]-1]) {
        time_derivative_add_value_gpu(time_deriv, int_data[2 + int_data[0] + int_data[1] + i_dep_var],rate*float_data[6+i_spec]);
      }
    }
  }
}

__device__
void rxn_gpu_troe_calc_deriv_contrib(ModelDataVariable *sc, TimeDerivativeGPU time_deriv, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double rate = rxn_env_data[0];
  for (int i_spec=0; i_spec<int_data[0]; i_spec++)
          rate *= sc->grid_cell_state[int_data[(2 + i_spec)]-1];
  if (rate!=ZERO) {
    int i_dep_var = 0;
    for (int i_spec=0; i_spec<int_data[0]; i_spec++, i_dep_var++) {
      if (int_data[(2 + int_data[0]+int_data[1]+i_dep_var)] < 0) continue;
      time_derivative_add_value_gpu(time_deriv, int_data[(2 + int_data[0]+int_data[1]+i_dep_var)], -rate);
    }
    for (int i_spec=0; i_spec<int_data[1]; i_spec++, i_dep_var++) {
      if (int_data[(2 + int_data[0]+int_data[1]+i_dep_var)] < 0) continue;
      if (-rate * float_data[(10 + i_spec)] * time_step <= sc->grid_cell_state[int_data[(2 + int_data[0]+ i_spec)]-1]) {
        time_derivative_add_value_gpu(time_deriv, int_data[(2 + int_data[0]+int_data[1]+i_dep_var)],rate*float_data[(10 + i_spec)]);
      }
    }
  }
}

__device__
void rxn_gpu_photolysis_calc_deriv_contrib(ModelDataVariable *sc, TimeDerivativeGPU time_deriv, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double rate = rxn_env_data[0];
  for (int i_spec=0; i_spec<int_data[0]; i_spec++)
          rate *= sc->grid_cell_state[int_data[(3 + i_spec)]-1];
  if (rate!=ZERO) {
    int i_dep_var = 0;
    for (int i_spec=0; i_spec<int_data[0]; i_spec++, i_dep_var++) {
      if (int_data[(3 + int_data[0]+int_data[1]+i_dep_var)] < 0) continue;
      time_derivative_add_value_gpu(time_deriv, int_data[(3 + int_data[0]+int_data[1]+i_dep_var)], -rate);
    }
    for (int i_spec=0; i_spec<int_data[1]; i_spec++, i_dep_var++) {
      if (int_data[(3 + int_data[0]+int_data[1]+i_dep_var)] < 0) continue;
        if (-rate * float_data[(1 + i_spec)] * time_step <= sc->grid_cell_state[int_data[(3 + int_data[0]+ i_spec)]-1]){
        time_derivative_add_value_gpu(time_deriv, int_data[(3 + int_data[0]+int_data[1]+i_dep_var)],rate*float_data[(1 + i_spec)]);
      }
    }
  }
}

#ifdef IS_DEBUG_MODE_removeAtomic
__device__
void jacobian_add_value_gpu(JacobianGPU jac, unsigned int elem_id,
                            int prod_or_loss,
                            double jac_contribution) {
  if (prod_or_loss == JACOBIAN_PRODUCTION) {
    jac.production_partials[elem_id] += jac_contribution;
  }
  else{ //(prod_or_loss == JACOBIAN_LOSS){
    jac.loss_partials[elem_id] += jac_contribution;
  }
}
#else
__device__
void jacobian_add_value_gpu(JacobianGPU jac, unsigned int elem_id,
                                   int prod_or_loss,
                                   double jac_contribution) {
  if (prod_or_loss == JACOBIAN_PRODUCTION) {
    atomicAdd_block(&(jac.production_partials[elem_id]), jac_contribution);
  }
  else{ //(prod_or_loss == JACOBIAN_LOSS){
    atomicAdd_block(&(jac.loss_partials[elem_id]),jac_contribution);
  }
}
#endif

__device__
void rxn_gpu_first_order_loss_calc_jac_contrib(ModelDataVariable *sc, JacobianGPU jac, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
if (int_data[3] >= 0) jacobian_add_value_gpu(jac, int_data[3], JACOBIAN_LOSS,
                                         rxn_env_data[0]);
}

__device__
void rxn_gpu_CMAQ_H2O2_calc_jac_contrib(ModelDataVariable *sc, JacobianGPU jac, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  int i_elem = 0;
  for (int i_ind = 0; i_ind < int_data[0]; i_ind++) {
    double rate = rxn_env_data[0];
    for (int i_spec = 0; i_spec < int_data[0]; i_spec++)
      if (i_ind != i_spec) rate *= sc->grid_cell_state[int_data[(2 + i_spec)]-1];
    for (int i_dep = 0; i_dep < int_data[0]; i_dep++, i_elem++) {
      if (int_data[(2 + 2*(int_data[0]+int_data[1])+i_elem)] < 0) continue;
      jacobian_add_value_gpu(jac, int_data[(2 + 2*(int_data[0]+int_data[1])+i_elem)], JACOBIAN_LOSS,
             rate);
    }
    for (int i_dep = 0; i_dep < int_data[1]; i_dep++, i_elem++) {
      if (int_data[(2 + 2*(int_data[0]+int_data[1])+i_elem)] < 0) continue;
      if (-rate * sc->grid_cell_state[int_data[(2 + i_ind)]-1] * float_data[(7 + i_dep)] * time_step <=
          sc->grid_cell_state[int_data[(2 + int_data[0]+ i_dep)]-1]) {
        jacobian_add_value_gpu(jac, int_data[(2 + 2*(int_data[0]+int_data[1])+i_elem)],
                   JACOBIAN_PRODUCTION, float_data[(7 + i_dep)] * rate);
      }
    }
  }
}


__device__
void rxn_gpu_CMAQ_OH_HNO3_calc_jac_contrib(ModelDataVariable *sc, JacobianGPU jac, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  int i_elem = 0;
  for (int i_ind = 0; i_ind < int_data[0]; i_ind++) {
    double rate = rxn_env_data[0];
    for (int i_spec = 0; i_spec < int_data[0]; i_spec++)
      if (i_ind != i_spec) rate *= sc->grid_cell_state[int_data[(2 + i_spec)]-1];
    for (int i_dep = 0; i_dep < int_data[0]; i_dep++, i_elem++) {
      if (int_data[(2 + 2*(int_data[0]+int_data[1]) + i_elem)] < 0) continue;
      jacobian_add_value_gpu(jac, int_data[(2 + 2*(int_data[0]+int_data[1]) + i_elem)], JACOBIAN_LOSS,
                   rate);
    }
    for (int i_dep = 0; i_dep < int_data[1]; i_dep++, i_elem++) {
      if (int_data[(2 + 2*(int_data[0]+int_data[1]) + i_elem)] < 0) continue;
      if (-rate * sc->grid_cell_state[int_data[(2 + i_ind)]-1] * float_data[(11 + i_dep)] * time_step <=
          sc->grid_cell_state[int_data[(2 + int_data[0] + i_dep)]-1]) {
        jacobian_add_value_gpu(jac, int_data[(2 + 2*(int_data[0]+int_data[1]) + i_elem)],
         JACOBIAN_PRODUCTION, float_data[(11 + i_dep)] * rate);
      }
    }
  }
}


__device__
void rxn_gpu_arrhenius_calc_jac_contrib(ModelDataVariable *sc, JacobianGPU jac, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  int i_elem = 0;
  for (int i_ind = 0; i_ind < int_data[0]; i_ind++) {
    double rate = rxn_env_data[0];
    for (int i_spec = 0; i_spec < int_data[0]; i_spec++)
      if (i_spec != i_ind) rate *= sc->grid_cell_state[int_data[(2 + i_spec)]-1];
    for (int i_dep = 0; i_dep < int_data[0]; i_dep++, i_elem++) {
      if (int_data[2 + 2*(int_data[0]+int_data[1]) + i_elem] < 0) continue;
      jacobian_add_value_gpu(jac, int_data[2 + 2*(int_data[0]+int_data[1]) + i_elem], JACOBIAN_LOSS,
                    rate);
    }
    for (int i_dep = 0; i_dep < int_data[1]; i_dep++, i_elem++) {
      if (int_data[2 + 2*(int_data[0]+int_data[1]) + i_elem] < 0) continue;
      if (-rate * sc->grid_cell_state[int_data[(2 + i_ind)]-1] * float_data[6+i_dep] * time_step <=
        sc->grid_cell_state[int_data[(2 + int_data[0] + i_dep)]-1]) {
        jacobian_add_value_gpu(jac, int_data[2 + 2*(int_data[0]+int_data[1]) + i_elem],
                           JACOBIAN_PRODUCTION, float_data[6+i_dep] * rate);
      }
    }
  }
}

__device__
void rxn_gpu_troe_calc_jac_contrib(ModelDataVariable *sc, JacobianGPU jac, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  int i_elem = 0;
  for (int i_ind = 0; i_ind < int_data[0]; i_ind++) {
    double rate = rxn_env_data[0];
    for (int i_spec = 0; i_spec < int_data[0]; i_spec++)
      if (i_ind != i_spec) rate *= sc->grid_cell_state[int_data[(2 + i_spec)]-1];
    for (int i_dep = 0; i_dep < int_data[0]; i_dep++, i_elem++) {
      if (int_data[(2 + 2*(int_data[0]+int_data[1])+i_elem)] < 0) continue;
        jacobian_add_value_gpu(jac, int_data[(2 + 2*(int_data[0]+int_data[1])+i_elem)], JACOBIAN_LOSS,
                   rate);
        }
    for (int i_dep = 0; i_dep < int_data[1]; i_dep++, i_elem++) {
      if (int_data[(2 + 2*(int_data[0]+int_data[1])+i_elem)] < 0) continue;
      if (-rate * sc->grid_cell_state[int_data[(2 + i_ind)]-1] * float_data[(10 + i_dep)] * time_step <=
        sc->grid_cell_state[int_data[(2 + int_data[0]+ i_dep)]-1]) {
        jacobian_add_value_gpu(jac, int_data[(2 + 2*(int_data[0]+int_data[1])+i_elem)], JACOBIAN_PRODUCTION,
                               float_data[(10 + i_dep)] * rate);
      }
    }
  }
}

__device__
void rxn_gpu_photolysis_calc_jac_contrib(ModelDataVariable *sc, JacobianGPU jac, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  int i_elem = 0;
  for (int i_ind = 0; i_ind < int_data[0]; i_ind++) {
    double rate = rxn_env_data[0];
    for (int i_spec = 0; i_spec < int_data[0]; i_spec++)
      if (i_spec != i_ind) rate *= sc->grid_cell_state[int_data[(3 + i_spec)]-1];
    for (int i_dep = 0; i_dep < int_data[0]; i_dep++, i_elem++) {
      if (int_data[(3 + 2*(int_data[0]+int_data[1])+i_elem)] < 0) continue;
      jacobian_add_value_gpu(jac, int_data[(3 + 2*(int_data[0]+int_data[1])+i_elem)], JACOBIAN_LOSS,
                   rate);
    }
    for (int i_dep = 0; i_dep < int_data[1]; i_dep++, i_elem++) {
      if (int_data[(3 + 2*(int_data[0]+int_data[1])+i_elem)] < 0) continue;
      if (-rate * sc->grid_cell_state[int_data[(3 + i_ind)]-1] * float_data[(1 + i_dep)] * time_step <=
          sc->grid_cell_state[int_data[(3 + int_data[0]+ i_dep)]-1]) {
      jacobian_add_value_gpu(jac, int_data[(3 + 2*(int_data[0]+int_data[1])+i_elem)],
              JACOBIAN_PRODUCTION, float_data[(1 + i_dep)] * rate);
      }
    }
  }
}

__device__ void cudaDevicemin_2(double *g_odata, double in, volatile double *sdata, int n_shr_empty){
  unsigned int tid = threadIdx.x;
  __syncthreads();
  sdata[tid] = in;
  __syncthreads();
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=sdata[tid];
  __syncthreads();
  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1)
  {
    if (tid < s){
      if(sdata[tid + s] < sdata[tid] ) sdata[tid]=sdata[tid + s];
    }
    __syncthreads();
  }
  *g_odata = sdata[0];
  __syncthreads();
}

#ifdef DEBUG_CVODE_GPU
__device__
void printmin(ModelDataGPU *md,double* y, const char *s) {
  __syncthreads();
  extern __shared__ double flag_shr2[];
  int tid= threadIdx.x + blockDim.x*blockIdx.x;
  __syncthreads();
  double min;
  cudaDevicemin_2(&min, y[tid], flag_shr2, md->n_shr_empty);
  __syncthreads();
  if(tid==0)printf("%s min %le\n",s,min);
  __syncthreads();
}
#endif

__device__ void cudaDeviceBCGprecond_2(double* dA, int* djA, int* diA, double* ddiag, double alpha){
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  int nnz=diA[blockDim.x];
  for(int j=diA[threadIdx.x];j<diA[threadIdx.x+1];j++){
    if(djA[j]==threadIdx.x){
      dA[j+nnz*blockIdx.x] = 1.0 + alpha*dA[j+nnz*blockIdx.x];
      if(dA[j+nnz*blockIdx.x]!=0.0){
        ddiag[row]= 1.0/dA[j+nnz*blockIdx.x];
       }else{
        ddiag[row]= 1.0;
      }
    }else{
      dA[j+nnz*blockIdx.x] = alpha*dA[j+nnz*blockIdx.x];
    }
  }
}

__device__ void cudaDeviceSpmv_2CSR(double* dx, double* db, double* dA, int* djA, int* diA){
  __syncthreads();
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  double sum = 0.0;
  int nnz=diA[blockDim.x];
  for(int j=diA[threadIdx.x]; j<diA[threadIdx.x+1]; j++){
    sum+= db[djA[j]+blockDim.x*blockIdx.x]*dA[j+nnz*blockIdx.x];
  }
  __syncthreads();
  dx[row]=sum;
  __syncthreads();
}

__device__ void cudaDeviceSpmv_2CSC_block(double* dx, double* db, double* dA, int* djA, int* diA){
  int row = threadIdx.x + blockDim.x*blockIdx.x;
  __syncthreads();
  dx[row]=0.0;
  __syncthreads();
  int nnz=diA[blockDim.x];
  for(int j=diA[threadIdx.x]; j<diA[threadIdx.x+1]; j++){
    double mult = db[row]*dA[j+nnz*blockIdx.x];
    atomicAdd_block(&(dx[djA[j]+blockDim.x*blockIdx.x]),mult);
  }
  __syncthreads();
}

__device__ void cudaDeviceSpmv_2(double* dx, double* db, double* dA, int* djA, int* diA){
#ifdef IS_DEBUG_MODE_CSR_ODE_GPU
  cudaDeviceSpmv_2CSR(dx,db,dA,djA,diA);
#else
  cudaDeviceSpmv_2CSC_block(dx,db,dA,djA,diA);
#endif
}

__device__ void warpReduce_2(volatile double *sdata, unsigned int tid) {
  unsigned int blockSize = blockDim.x;
  if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
  if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
  if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
  if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
  if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
  if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

__device__ void cudaDevicedotxy_2(double *g_idata1, double *g_idata2,
                                 double *g_odata, int n_shr_empty){
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  __syncthreads();
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=0.;
  __syncthreads();
#ifdef IS_DEBUG_MODE_cudaDevicedotxy_2
  //used for compare with cpu
  sdata[0]=0.;
  __syncthreads();
  if(tid==0){
    for(int j=0;j<blockDim.x;j++){
      sdata[0]+=g_idata1[j+blockIdx.x*blockDim.x]*g_idata2[j+blockIdx.x*blockDim.x];
    }
  }
#else
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  sdata[tid] = g_idata1[i]*g_idata2[i];
  __syncthreads();
  unsigned int blockSize = blockDim.x+n_shr_empty;
  if ((blockSize >= 1024) && (tid < 512)) {
    sdata[tid] += sdata[tid + 512];
  }
  __syncthreads();
  if ((blockSize >= 512) && (tid < 256)) {
    sdata[tid] += sdata[tid + 256];
  }
  __syncthreads();
  if ((blockSize >= 256) && (tid < 128)) {
    sdata[tid] += sdata[tid + 128];
  }
  __syncthreads();
  if ((blockSize >= 128) && (tid < 64)) {
    sdata[tid] += sdata[tid + 64];
  }
  __syncthreads();
  if (tid < 32) warpReduce_2(sdata, tid);
#endif
  __syncthreads();
  *g_odata = sdata[0];
  __syncthreads();
}

__device__ void cudaDeviceVWRMS_Norm_2(double *g_idata1, double *g_idata2, double *g_odata, int n_shr_empty){
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  __syncthreads();
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=0.;
  __syncthreads();
  sdata[tid] = g_idata1[i]*g_idata2[i];
  sdata[tid] = sdata[tid]*sdata[tid];
  __syncthreads();
#ifdef DEBUG_cudaDevicedotxy_2
  //used for compare with cpu
  if(tid==0){
    double sum=0.;
    for(int j=0;j<blockDim.x;j++){
      sum+=sdata[j];
    }
    sdata[0]=sum;
  }
  __syncthreads();
#else
  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1){
    if (tid < s)
      sdata[tid] += sdata[tid + s];
    __syncthreads();
  }
#endif
  g_odata[0] = sqrt(sdata[0]/blockDim.x);
  __syncthreads();
}

__device__
void cudaDeviceJacCopy(int* Ap, double* Ax, double* Bx) {
  __syncthreads();
  int nnz=Ap[blockDim.x];
  for(int j=Ap[threadIdx.x]; j<Ap[threadIdx.x+1]; j++){
    Bx[j+nnz*blockIdx.x]=Ax[j+nnz*blockIdx.x];
  }
  __syncthreads();
}

__device__
int cudaDevicecamp_solver_check_model_state(ModelDataGPU *md, ModelDataVariable *sc, double *y, int *flag)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  __syncthreads();
  extern __shared__ int flag_shr[];
  flag_shr[0] = 0;
  __syncthreads();
  if (y[i] < -SMALL) {
    flag_shr[0] = CAMP_SOLVER_FAIL;
  } else {
    md->state[md->map_state_deriv[i]] =
            y[i] <= -SMALL ?
            TINY : y[i];
  }
  __syncthreads();
  *flag = flag_shr[0];
  __syncthreads();
  return *flag;
}

__device__ void solveRXN(
  int i_rxn,TimeDerivativeGPU deriv_data,
  double time_step,ModelDataGPU *md, ModelDataVariable *sc
){
  double *rxn_float_data = (double *)&( md->rxn_double[md->rxn_float_indices[i_rxn]]);
  int *int_data = (int *)&(md->rxn_int[md->rxn_int_indices[i_rxn]]);
  int *rxn_int_data = (int *) &(int_data[1]);
  double *rxn_env_data = &(md->rxn_env_data
  [md->n_rxn_env_data*blockIdx.x+md->rxn_env_data_idx[i_rxn]]);
  switch (int_data[0]) {
    case RXN_ARRHENIUS :
      rxn_gpu_arrhenius_calc_deriv_contrib(sc, deriv_data, rxn_int_data,
                                           rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_CMAQ_H2O2 :
      rxn_gpu_CMAQ_H2O2_calc_deriv_contrib(sc, deriv_data, rxn_int_data,
                                           rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_CMAQ_OH_HNO3 :
      rxn_gpu_CMAQ_OH_HNO3_calc_deriv_contrib(sc, deriv_data, rxn_int_data,
                                              rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_FIRST_ORDER_LOSS:
    rxn_gpu_first_order_loss_calc_deriv_contrib(sc, deriv_data, rxn_int_data,
                                    rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_PHOTOLYSIS :
      rxn_gpu_photolysis_calc_deriv_contrib(sc, deriv_data, rxn_int_data,
                                            rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_TROE :
      rxn_gpu_troe_calc_deriv_contrib(sc, deriv_data, rxn_int_data,
                                      rxn_float_data, rxn_env_data,time_step);
      break;
  }
}

__device__ void cudaDevicecalc_deriv(double time_step, double *y,
        double *yout, ModelDataGPU *md, ModelDataVariable *sc)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  md->J_tmp[i]=y[i]-md->J_state[i];
  cudaDeviceSpmv_2(md->J_tmp2, md->J_tmp, md->J_solver, md->djA, md->diA);
  md->J_tmp[i]=md->J_deriv[i]+md->J_tmp2[i];
  TimeDerivativeGPU deriv_data;
  deriv_data.production_rates = md->production_rates;
  deriv_data.loss_rates = md->loss_rates;
  __syncthreads();
  deriv_data.production_rates[i] = 0.0;
  deriv_data.loss_rates[i] = 0.0;
  __syncthreads();
  deriv_data.production_rates = &( md->production_rates[blockDim.x*blockIdx.x]);
  deriv_data.loss_rates = &( md->loss_rates[blockDim.x*blockIdx.x]);
  sc->grid_cell_state = &( md->state[md->state_size_cell*blockIdx.x]);
  int n_rxn = md->n_rxn;
  __syncthreads();
#ifdef IS_DEBUG_MODE_removeAtomic
  if(threadIdx.x==0){
    for (int j = 0; j < n_rxn; j++){
      //printf("n_rxn %d i %d j %d \n",n_rxn,i,j);
      solveRXN(j,deriv_data, time_step, md, sc);
    }
  }
#else
  if( threadIdx.x < n_rxn) {
    int n_iters = n_rxn / blockDim.x;
    for (int j = 0; j < n_iters; j++) {
      int i_rxn = threadIdx.x + j*blockDim.x;
      solveRXN(i_rxn,deriv_data, time_step, md, sc);
    }
    int residual=n_rxn%blockDim.x;
    if(threadIdx.x < residual){
      int i_rxn = threadIdx.x + blockDim.x*n_iters;
      solveRXN(i_rxn, deriv_data, time_step, md, sc);
    }
  }
#endif
  __syncthreads();
  deriv_data.production_rates = md->production_rates;
  deriv_data.loss_rates = md->loss_rates;
  __syncthreads();
  double *r_p = deriv_data.production_rates;
  double *r_l = deriv_data.loss_rates;
  if (r_p[i] + r_l[i] != 0.0) {
    if (md->use_deriv_est==1) {
      double scale_fact = 1.0 / (r_p[i] + r_l[i]) /
          (1.0 / (r_p[i] + r_l[i]) + MAX_PRECISION_LOSS / fabs(r_p[i]- r_l[i]));
      yout[i] = scale_fact * (r_p[i] - r_l[i]) + (1.0 - scale_fact) * (md->J_tmp[i]);
    }else {
      yout[i] = r_p[i] - r_l[i];
    }
  } else {
    yout[i] = 0.0;
  }
  //print_double(y,86,"y646");
  //print_double(md->J_state,86,"J_state644");
  //print_double(md->J_tmp,86,"J_tmp643");
  //print_double(md->J_deriv,86,"J_deriv644");
  //print_double(md->J_tmp2,86,"J_tmp2645");
  //print_double(deriv_data.loss_rates,86,"loss_rates");
  //print_double(deriv_data.production_rates,86,"production_rates");
  //print_double(yout,86,"deriv_data645");
  __syncthreads();
}

__device__
int cudaDevicef(double time_step, double *y,
        double *yout, ModelDataGPU *md, ModelDataVariable *sc, int *flag)
{
  __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  int clock_khz=md->clock_khz;
  clock_t start;
  start = clock();
#endif
  time_step = sc->cv_next_h;
  time_step = time_step > 0. ? time_step : md->init_time_step;
  //print_double(md->state,md->state_size_cell,"state661");
  int checkflag=cudaDevicecamp_solver_check_model_state(md, sc, y, flag);
  //print_double(md->state,md->state_size_cell,"state663");
  __syncthreads();
  if(checkflag==CAMP_SOLVER_FAIL){
    *flag=CAMP_SOLVER_FAIL;
    __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadIdx.x==0) sc->timef += ((double)(int)(clock() - start))/(clock_khz*1000);
#endif
    return CAMP_SOLVER_FAIL;
  }
  cudaDevicecalc_deriv(time_step, y, yout, md, sc);
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadIdx.x==0) sc->timef += ((double)(int)(clock() - start))/(clock_khz*1000);
#endif
  __syncthreads();
  *flag=0;
  __syncthreads();
  return 0;
}

__device__
int CudaDeviceguess_helper(double h_n, double* y_n,
   double* y_n1, double* hf, double* atmp1,
   double* acorr, int *flag, ModelDataGPU *md, ModelDataVariable *sc
) {
  extern __shared__ double sdata[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  __syncthreads();
  double min;
  cudaDevicemin_2(&min, y_n[i], sdata, md->n_shr_empty);
  if(min>-SMALL){
    return 0;
  }
  __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  int clock_khz=md->clock_khz;
  clock_t start;
  start = clock();
#endif
  atmp1[i]=y_n1[i];
  __syncthreads();
  if (h_n > 0.) {
    acorr[i]=(1./h_n)*hf[i];
  } else {
    acorr[i]=hf[i];
  }
  //print_double(acorr,86,"acorr711");
  double t_0 = h_n > 0. ? sc->cv_tn - h_n : sc->cv_tn - 1.;
  double t_j = 0.;
  __syncthreads();
  for (int iter = 0; iter < GUESS_MAX_ITER && t_0 + t_j < sc->cv_tn; iter++) {
    __syncthreads();
    double h_j = sc->cv_tn - (t_0 + t_j);
    //print_double(atmp1,86,"atmp720");
#ifdef IS_DEBUG_MODE_CudaDeviceguess_helper
    if(threadIdx.x==0){
      int i_fast = -1;
      for (int j = 0; j < blockDim.x; j++) {
        double t_star = -atmp1[j+blockIdx.x*blockDim.x] / acorr[j+blockIdx.x*blockDim.x];
        if ((t_star > 0. || (t_star == 0. && acorr[j+blockIdx.x*blockDim.x] < 0.)) &&
            t_star < h_j) {
          h_j = t_star;
          i_fast = 1;
        }
      }
      if (i_fast >= 0 && h_n > 0.)
        h_j *= 0.95 + 0.1 * iter / (double)GUESS_MAX_ITER;
      sdata[0]=h_j;
    }
    __syncthreads();
    h_j=sdata[0];
    __syncthreads();
#else
    double t_star = -atmp1[i] / acorr[i];
    if (t_star < 0. || (t_star == 0. && acorr[i] >= 0.)){
      t_star=h_j;
    }
    cudaDevicemin_2(&min, t_star, sdata, md->n_shr_empty);
    if(min<h_j){
      h_j = min;
      h_j *= 0.95 + 0.1 * iter / (double)GUESS_MAX_ITER;
    }
#endif
    h_j = sc->cv_tn < t_0 + t_j + h_j ? sc->cv_tn - (t_0 + t_j) : h_j;
    __syncthreads();
    if (h_n == 0. && sc->cv_tn - (h_j + t_j + t_0) > md->cv_reltol) {
      __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0) sc->timeguess_helper += ((double)(clock() - start))/(clock_khz*1000);
#endif
    return -1;
    }
    atmp1[i]+=h_j*acorr[i];
    __syncthreads();
    t_j += h_j;
    int aux_flag=0;
    //print_double(atmp1,86,"atmp1766");
    int fflag=cudaDevicef(t_0 + t_j, atmp1, acorr,md,sc,&aux_flag);
    //print_double(acorr,86,"acorr721");
    __syncthreads();
    if (fflag == CAMP_SOLVER_FAIL) {
      acorr[i] = 0.;
      __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0) sc->timeguess_helper += ((double)(clock() - start))/(clock_khz*1000);
#endif
     return -1;
    }
    if (iter == GUESS_MAX_ITER - 1 && t_0 + t_j < sc->cv_tn) {
      if (h_n == 0.){
        __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0) sc->timeguess_helper += ((double)(clock() - start))/(clock_khz*1000);
#endif
        return -1;
      }
    }
    __syncthreads();
  }
  __syncthreads();
  acorr[i]=atmp1[i]-y_n[i];
  if (h_n > 0.) acorr[i]=acorr[i]*0.999;
  hf[i]=atmp1[i]-y_n1[i];
  __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0)  sc->timeguess_helper += ((double)(clock() - start))/(clock_khz*1000);
#endif
  __syncthreads();
  return 1;
}

__device__ void solveRXNJac(
        int i_rxn, JacobianGPU jac,
        ModelDataGPU *md, ModelDataVariable *sc
){
  double *rxn_float_data = (double *)&( md->rxn_double[md->rxn_float_indices[i_rxn]]);
  int *int_data = (int *)&(md->rxn_int[md->rxn_int_indices[i_rxn]]);
  int *rxn_int_data = (int *) &(int_data[1]);
  double *rxn_env_data = &(md->rxn_env_data
  [md->n_rxn_env_data*blockIdx.x+md->rxn_env_data_idx[i_rxn]]);
  switch (int_data[0]) {
    case RXN_ARRHENIUS :
      rxn_gpu_arrhenius_calc_jac_contrib(sc, jac, rxn_int_data,
                                         rxn_float_data, rxn_env_data,sc->cv_next_h);
      break;
    case RXN_CMAQ_H2O2 :
      rxn_gpu_CMAQ_H2O2_calc_jac_contrib(sc, jac, rxn_int_data,
                                         rxn_float_data, rxn_env_data,sc->cv_next_h);
      break;
    case RXN_CMAQ_OH_HNO3 :
      rxn_gpu_CMAQ_OH_HNO3_calc_jac_contrib(sc, jac, rxn_int_data,
                                            rxn_float_data, rxn_env_data,sc->cv_next_h);
      break;
    case RXN_FIRST_ORDER_LOSS :
      rxn_gpu_first_order_loss_calc_jac_contrib(sc, jac, rxn_int_data,
                                        rxn_float_data, rxn_env_data,sc->cv_next_h);
      break;
    case RXN_PHOTOLYSIS :
      rxn_gpu_photolysis_calc_jac_contrib(sc, jac, rxn_int_data,
                                          rxn_float_data, rxn_env_data,sc->cv_next_h);
      break;
    case RXN_TROE :
      rxn_gpu_troe_calc_jac_contrib(sc, jac, rxn_int_data,
                                    rxn_float_data, rxn_env_data,sc->cv_next_h);
      break;
  }
}

__device__ void cudaDevicecalc_Jac(double *y,ModelDataGPU *md, ModelDataVariable *sc
){
  __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  int clock_khz=md->clock_khz;
  clock_t start;
  start = clock();
  __syncthreads();
#endif
  JacobianGPU *jac = &md->jac;
  JacobianGPU jacBlock;
  __syncthreads();
  jacBlock.num_elem = jac->num_elem;
  jacBlock.production_partials = &( jac->production_partials[jacBlock.num_elem[0]*blockIdx.x]);
  jacBlock.loss_partials = &( jac->loss_partials[jacBlock.num_elem[0]*blockIdx.x]);
  __syncthreads();
  sc->grid_cell_state = &( md->state[md->state_size_cell*blockIdx.x]);
  __syncthreads();
  int n_rxn = md->n_rxn;
#ifdef IS_DEBUG_MODE_removeAtomic
  if(threadIdx.x==0){
    for (int j = 0; j < n_rxn; j++){
      solveRXNJac(j,jacBlock, md, sc);
    }
  }
#else
  if( threadIdx.x < n_rxn) {
    int n_iters = n_rxn / blockDim.x;
    for (int j = 0; j < n_iters; j++) {
      int i_rxn = threadIdx.x + j*blockDim.x;
      solveRXNJac(i_rxn,jacBlock, md, sc);
    }
    int residual=n_rxn%blockDim.x;
    if(threadIdx.x < residual){
      int i_rxn = threadIdx.x + blockDim.x*n_iters;
      solveRXNJac(i_rxn,jacBlock, md, sc);
    }
  }
#endif
  __syncthreads();
  JacMap *jac_map = md->jac_map;
  int nnz = md->n_mapped_values[0];
  int n_iters = nnz / blockDim.x;
  for (int z = 0; z < n_iters; z++) {
    int j = threadIdx.x + z*blockDim.x;
    md->dA[jac_map[j].solver_id + nnz * blockIdx.x] =
    jacBlock.production_partials[jac_map[j].rxn_id] - jacBlock.loss_partials[jac_map[j].rxn_id];
    jacBlock.production_partials[jac_map[j].rxn_id] = 0.0;
    jacBlock.loss_partials[jac_map[j].rxn_id] = 0.0;
  }
  int residual=nnz%blockDim.x;
  if(threadIdx.x < residual){
    int j = threadIdx.x + n_iters*blockDim.x;
  md->dA[jac_map[j].solver_id + nnz * blockIdx.x] =
      jacBlock.production_partials[jac_map[j].rxn_id] - jacBlock.loss_partials[jac_map[j].rxn_id];
    jacBlock.production_partials[jac_map[j].rxn_id] = 0.0;
    jacBlock.loss_partials[jac_map[j].rxn_id] = 0.0;
  }
    __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0) sc->timecalc_Jac += ((double)(clock() - start))/(clock_khz*1000);
#endif
}

__device__
int cudaDeviceJac(int *flag, ModelDataGPU *md, ModelDataVariable *sc)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int retval;
  __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  int clock_khz=md->clock_khz;
  clock_t start;
  start = clock();
#endif
  md->use_deriv_est=0;
  int aux_flag=0;
  __syncthreads();
  //print_double(md->dcv_y,86,"dcv_y914");
  retval=cudaDevicef(sc->cv_next_h, md->dcv_y, md->dftemp,md,sc,&aux_flag);
  md->use_deriv_est=1;
  __syncthreads();
  if(retval==CAMP_SOLVER_FAIL)
    return CAMP_SOLVER_FAIL;
  cudaDevicecalc_Jac(md->dcv_y,md, sc);
  __syncthreads();
  int nnz = md->n_mapped_values[0];
  int n_iters = nnz / blockDim.x;
  for (int z = 0; z < n_iters; z++) {
    int j = threadIdx.x + z*blockDim.x + nnz * blockIdx.x;
    md->J_solver[j]=md->dA[j];
  }
  int residual=nnz%blockDim.x;
  if(threadIdx.x < residual){
    int j = threadIdx.x + n_iters*blockDim.x + nnz * blockIdx.x;
    md->J_solver[j]=md->dA[j];
  }
  __syncthreads();
  md->J_state[i]=md->dcv_y[i];
  md->J_deriv[i]=md->dftemp[i];
  __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0)  sc->timeJac += ((double)(clock() - start))/(clock_khz*1000);
#endif
  __syncthreads();
  *flag = 0;
  __syncthreads();
  return 0;
}

__device__
int cudaDevicelinsolsetup(
    ModelDataGPU *md, ModelDataVariable *sc, int convfail
) {
  extern __shared__ int flag_shr[];
  double dgamma;
  int jbad, jok;
  dgamma = fabs((sc->cv_gamma / sc->cv_gammap) - 1.);//SUNRabs
  jbad = (sc->cv_nst == 0) ||
         (sc->cv_nst > sc->nstlj + CVD_MSBJ) ||
         ((convfail == CV_FAIL_BAD_J) && (dgamma < CVD_DGMAX)) ||
         (convfail == CV_FAIL_OTHER);
  jok = !jbad;
  if (jok==1) {
    __syncthreads();
    sc->cv_jcur = 0;
    cudaDeviceJacCopy(md->diA, md->dsavedJ, md->dA);
    __syncthreads();
  } else {
    __syncthreads();
    sc->nstlj = sc->cv_nst;
    sc->cv_jcur = 1;
    __syncthreads();
    int aux_flag=0;
    __syncthreads();
    int guess_flag=cudaDeviceJac(&aux_flag,md,sc);
    __syncthreads();
    if (guess_flag < 0) {
      return -1;}
    if (guess_flag > 0) {
      return 1;}
   cudaDeviceJacCopy(md->diA, md->dA, md->dsavedJ);
  }
  __syncthreads();
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  md->dx[i]=0.;
  cudaDeviceBCGprecond_2(md->dA, md->djA, md->diA, md->ddiag, -sc->cv_gamma);
  __syncthreads();
  return 0;
}

__device__
void solveBcgCudaDeviceCVODE(ModelDataGPU *md, ModelDataVariable *sc)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double alpha,rho0,omega0,beta,rho1,temp1,temp2;
  alpha=rho0=omega0=beta=rho1=temp1=temp2=1.0;
  md->dn0[i]=0.0;
  md->dp0[i]=0.0;
  cudaDeviceSpmv_2(md->dr0,md->dx,md->dA,md->djA,md->diA);
  md->dr0[i]=md->dtempv[i]-md->dr0[i];
  md->dr0h[i]=md->dr0[i];
  int it=0;
  while(it<1000 && temp1>1.0E-30){
    cudaDevicedotxy_2(md->dr0, md->dr0h, &rho1, md->n_shr_empty);
    beta = (rho1 / rho0) * (alpha / omega0);
    md->dp0[i]=beta*md->dp0[i]+md->dr0[i]-md->dn0[i]*omega0*beta;
    md->dy[i]=md->ddiag[i]*md->dp0[i];
    cudaDeviceSpmv_2(md->dn0, md->dy, md->dA, md->djA, md->diA);
    cudaDevicedotxy_2(md->dr0h, md->dn0, &temp1, md->n_shr_empty);
    alpha = rho1 / temp1;
    md->ds[i]=md->dr0[i]-alpha*md->dn0[i];
    md->dx[i]+=alpha*md->dy[i];
    md->dy[i]=md->ddiag[i]*md->ds[i];
    cudaDeviceSpmv_2(md->dt, md->dy, md->dA, md->djA, md->diA);
    md->dr0[i]=md->ddiag[i]*md->dt[i];
    cudaDevicedotxy_2(md->dy, md->dr0, &temp1, md->n_shr_empty);
    cudaDevicedotxy_2(md->dr0, md->dr0, &temp2, md->n_shr_empty);
    omega0 = temp1 / temp2;
    md->dx[i]+=omega0*md->dy[i];
    md->dr0[i]=md->ds[i]-omega0*md->dt[i];
    md->dt[i]=0.0;
    cudaDevicedotxy_2(md->dr0, md->dr0, &temp1, md->n_shr_empty);
    temp1 = sqrt(temp1);
    rho0 = rho1;
    it++;
  __syncthreads();
  //if(i==0)printf("end iter %d BCG GPU\n",it);
  }
  __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  if(threadIdx.x==0) sc->counterBCGInternal += it;
  if(threadIdx.x==0) sc->counterBCG++;
#endif
}

__device__
int cudaDevicecvNewtonIteration(ModelDataGPU *md, ModelDataVariable *sc){
  extern __shared__ double flag_shr2[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int aux_flag=0;
  double del, delp, dcon;
  int m = 0;
  del = delp = 0.0;
  int retval;
  __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  int clock_khz=md->clock_khz;
  clock_t start;
#endif
  for(;;) {
    __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    start = clock();
#endif
    //print_double(md->dtempv,86,"dtempvN_VLinearSum1");
    md->dtempv[i]=sc->cv_rl1*md->dzn[i+md->nrows]+md->cv_acor[i];
    //print_double(md->dtempv,86,"dtempvN_VLinearSum2");
    md->dtempv[i]=sc->cv_gamma*md->dftemp[i]-md->dtempv[i];
    //print_double(md->dtempv,86,"dtempvcv_lsolve1");
    solveBcgCudaDeviceCVODE(md, sc);
    __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0) sc->dtBCG += ((double)(int)(clock() - start))/(clock_khz*1000);
#endif
    md->dtempv[i] = md->dx[i];
    //print_double(md->dtempv,86,"dtempvcv_lsolve2");
    __syncthreads();
    cudaDeviceVWRMS_Norm_2(md->dx, md->dewt, &del, md->n_shr_empty);
    md->dftemp[i]=md->dcv_y[i]+md->dtempv[i];
    __syncthreads();
    //print_double(md->dcv_y,86,"dcv_y2994");
    //print_double(md->dftemp,86,"cv_ftemplsolve");
    int guessflag=CudaDeviceguess_helper(0., md->dftemp,
       md->dcv_y, md->dtempv, md->dtempv1,md->dtempv2, &aux_flag, md, sc
    );
    __syncthreads();
    if (guessflag < 0) {
      if (!(sc->cv_jcur)) { //Bool set up during linsolsetup just before Jacobian
        return TRY_AGAIN;
      } else {
        return RHSFUNC_RECVR;
      }
    }
    md->dftemp[i]=md->dcv_y[i]+md->dtempv[i];
    double min;
    cudaDevicemin_2(&min, md->dftemp[i], flag_shr2, md->n_shr_empty);
    if (min < -CAMP_TINY) {
      return CONV_FAIL;
    }
    __syncthreads();
    md->cv_acor[i]+=md->dtempv[i];
    md->dcv_y[i]=md->dzn[i]+md->cv_acor[i];
    //print_double(md->cv_acor,86,"cv_acor1060");
    //print_double(md->dcv_y,86,"dcv_y1060");
    if (m > 0) {
      sc->cv_crate = SUNMAX(0.3 * sc->cv_crate, del / delp);
    }
    dcon = del * SUNMIN(1.0, sc->cv_crate) / md->cv_tq[4+blockIdx.x*(NUM_TESTS + 1)];
    flag_shr2[0]=0;
    __syncthreads();
    if (dcon <= 1.) {
      //print_double(md->cv_acor,86,"cv_acor1505");
      //print_double(md->dewt,86,"dewt1505");
      cudaDeviceVWRMS_Norm_2(md->cv_acor, md->dewt, &sc->cv_acnrm, md->n_shr_empty);
      //print_double(&sc->cv_acnrm,1,"cv_acnrm1151");
      __syncthreads();
      sc->cv_jcur = 0;
      __syncthreads();
      return CV_SUCCESS;
    }
    m++;
    if ((m == md->cv_maxcor) || ((m >= 2) && (del > RDIV * delp))) {
      if (!(sc->cv_jcur)) {
        return TRY_AGAIN;
      } else {
        return RHSFUNC_RECVR;
      }
    }
    delp = del;
    __syncthreads();
    //print_double(md->dcv_y,86,"dcv_y1137");
    retval=cudaDevicef(sc->cv_next_h, md->dcv_y, md->dftemp, md, sc, &aux_flag);
    __syncthreads();
    md->cv_acor[i]=md->dcv_y[i]+md->dzn[i];
    if (retval < 0) {
      return CV_RHSFUNC_FAIL;
    }
    if (retval > 0) {
      if (!(sc->cv_jcur)) {
        return TRY_AGAIN;
      } else {
        return RHSFUNC_RECVR;
      }
    }
    __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0) sc->dtPostBCG += ((double)(clock() - start))/(clock_khz*1000);
#endif
  }
}

__device__
int cudaDevicecvNlsNewton(int nflag,
        ModelDataGPU *md, ModelDataVariable *sc
) {
  extern __shared__ int flag_shr[];
  int flagDevice = 0;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int retval=0;
  __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  int clock_khz=md->clock_khz;
  clock_t start;
#endif
  int convfail = ((nflag == FIRST_CALL) || (nflag == PREV_ERR_FAIL)) ?
                 CV_NO_FAILURES : CV_FAIL_OTHER;
  int dgamrat=fabs(sc->cv_gamrat - 1.);
  int callSetup = (nflag == PREV_CONV_FAIL) || (nflag == PREV_ERR_FAIL) ||
                  (sc->cv_nst == 0) ||
                  (sc->cv_nst >= sc->cv_nstlp + MSBP) ||
                  (dgamrat > DGMAX);
  __syncthreads();
  //print_double(md->cv_last_yn,86,"cv_last_yn1175");
  md->dftemp[i]=md->dzn[i]-md->cv_last_yn[i];
  //print_double(md->dftemp,86,"cv_ftemppN_VLinearSum2");
  md->cv_acor_init[i]=0.;
  __syncthreads();
  int guessflag=CudaDeviceguess_helper(sc->cv_h, md->dzn,
       md->cv_last_yn, md->dftemp, md->dtempv1,
       md->cv_acor_init,  &flagDevice,md, sc
  );
  __syncthreads();
  if(guessflag<0){
    return RHSFUNC_RECVR;
  }
  for(;;) {
    __syncthreads();
    //print_double(md->cv_acor_init,86,"cv_acor_init1140");
    md->dcv_y[i] = md->dzn[i]+md->cv_acor_init[i];
    //print_double(md->dcv_y,86,"dcv_y1139");
    int aux_flag=0;
    //print_double(md->dftemp,86,"cv_ftemppcv_f1");
    //print_double(&sc->cv_tn,1,"cv_tn1216");
    retval=cudaDevicef(sc->cv_tn, md->dcv_y,md->dftemp,md,sc,&aux_flag);
    //print_double(md->dftemp,86,"cv_ftemppcv_f2");
    //print_double(md->dcv_y,86,"dcv_y1144");
    if (retval < 0) {
      return CV_RHSFUNC_FAIL;
    }
    if (retval> 0) {
      return RHSFUNC_RECVR;
    }
    if (callSetup) {
      __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
      start = clock();
#endif
      __syncthreads();
      int linflag=cudaDevicelinsolsetup(md, sc,convfail);
      //print_double(md->dftemp,86,"cv_ftempp1160");
      __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
      if(threadIdx.x==0) sc->timelinsolsetup += ((double)(clock() - start))/(clock_khz*1000);
#endif
      callSetup = 0;
      sc->cv_gamrat = sc->cv_crate = 1.0;
      sc->cv_gammap = sc->cv_gamma;
      sc->cv_nstlp = sc->cv_nst;
      __syncthreads();
      if (linflag < 0) {
        flag_shr[0] = CV_LSETUP_FAIL;
        break;
      }
      if (linflag > 0) {
        flag_shr[0] = CONV_FAIL;
        break;
      }
    }
    __syncthreads();
    md->cv_acor[i] = md->cv_acor_init[i];
    __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    start = clock();
#endif
    __syncthreads();
    int nItflag=cudaDevicecvNewtonIteration(md, sc);
    __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0)  sc->timeNewtonIteration += ((double)(clock() - start))/(clock_khz*1000);
#endif
    if (nItflag != TRY_AGAIN) {
      return nItflag;
    }
    __syncthreads();
    callSetup = 1;
    __syncthreads();
    convfail = CV_FAIL_BAD_J;
    __syncthreads();
  } //for(;;)
  __syncthreads();
  return nflag;
}

__device__
void cudaDevicecvRescale(ModelDataGPU *md, ModelDataVariable *sc) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double factor;
  __syncthreads();
  factor = sc->cv_eta;
  for (int j=1; j <= sc->cv_q; j++) {
    md->dzn[i+md->nrows*j]*=factor;
    __syncthreads();
    factor *= sc->cv_eta;
    __syncthreads();
  }
  //print_double(&sc->cv_eta,1,"cv_eta_1290");
  sc->cv_h = sc->cv_hscale * sc->cv_eta;
  sc->cv_next_h = sc->cv_h;
  sc->cv_hscale = sc->cv_h;
  __syncthreads();
}

__device__
void cudaDevicecvRestore(ModelDataGPU *md, ModelDataVariable *sc, double saved_t) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j, k;
  __syncthreads();
  sc->cv_tn=saved_t;
  print_double(md->dzn,86,"dzn1299");
  for (k = 1; k <= sc->cv_q; k++){
    for (j = sc->cv_q; j >= k; j--) {
      md->dzn[i+md->nrows*(j-1)]-=md->dzn[i+md->nrows*j];
    }
  }
  md->dzn[i]=md->cv_last_yn[i];
  print_double(md->dzn,86,"dzn1306");
  __syncthreads();
}

__device__
int cudaDevicecvHandleNFlag(ModelDataGPU *md, ModelDataVariable *sc, int *nflagPtr, double saved_t,
                             int *ncfPtr) {
  extern __shared__ int flag_shr[];
  if (*nflagPtr == CV_SUCCESS){
    return(DO_ERROR_TEST);
  }
  cudaDevicecvRestore(md, sc, saved_t);
  if (*nflagPtr == CV_LSETUP_FAIL)  return(CV_LSETUP_FAIL);
  if (*nflagPtr == CV_LSOLVE_FAIL)  return(CV_LSOLVE_FAIL);
  if (*nflagPtr == CV_RHSFUNC_FAIL) return(CV_RHSFUNC_FAIL);
  (*ncfPtr)++;
  sc->cv_etamax = 1.;
  __syncthreads();
  if ((fabs(sc->cv_h) <= sc->cv_hmin*ONEPSM) ||
      (*ncfPtr == sc->cv_maxncf)) {
    if (*nflagPtr == CONV_FAIL)     return(CV_CONV_FAILURE);
    if (*nflagPtr == RHSFUNC_RECVR) return(CV_REPTD_RHSFUNC_ERR);
  }
  __syncthreads();
  sc->cv_eta = SUNMAX(ETACF,
          sc->cv_hmin / fabs(sc->cv_h));
  //print_double(&sc->cv_eta,1,"cv_eta_1337");
  __syncthreads();
  *nflagPtr = PREV_CONV_FAIL;
  cudaDevicecvRescale(md, sc);
  __syncthreads();
  return (PREDICT_AGAIN);
}

__device__
void cudaDevicecvSetTqBDFt(ModelDataGPU *md, ModelDataVariable *sc,
                           double hsum, double alpha0, double alpha0_hat,
                           double xi_inv, double xistar_inv) {
  extern __shared__ int flag_shr[];
  double A1, A2, A3, A4, A5, A6;
  double C, Cpinv, Cppinv;
  __syncthreads();
  A1 = 1. - alpha0_hat + alpha0;
  A2 = 1. + sc->cv_q * A1;
  md->cv_tq[2+blockIdx.x*(NUM_TESTS + 1)] = fabs(A1 / (alpha0 * A2));
  md->cv_tq[5+blockIdx.x*(NUM_TESTS + 1)] = fabs(A2 * xistar_inv / (md->cv_l[sc->cv_q+blockIdx.x*L_MAX] * xi_inv));
  if (sc->cv_qwait == 1) {
    if (sc->cv_q > 1) {
      C = xistar_inv / md->cv_l[sc->cv_q+blockIdx.x*L_MAX];
      A3 = alpha0 + 1. / sc->cv_q;
      A4 = alpha0_hat + xi_inv;
      Cpinv = (1. - A4 + A3) / A3;
      md->cv_tq[1+blockIdx.x*(NUM_TESTS + 1)] = fabs(C * Cpinv);
    }
    else md->cv_tq[1+blockIdx.x*(NUM_TESTS + 1)] = 1.;
    __syncthreads();
    hsum += md->cv_tau[sc->cv_q+blockIdx.x*(L_MAX + 1)];
    xi_inv = sc->cv_h / hsum;
    A5 = alpha0 - (1. / (sc->cv_q+1));
    A6 = alpha0_hat - xi_inv;
    Cppinv = (1. - A6 + A5) / A2;
    md->cv_tq[3+blockIdx.x*(NUM_TESTS + 1)] = fabs(Cppinv / (xi_inv * (sc->cv_q+2) * A5));
    __syncthreads();
  }
  md->cv_tq[4+blockIdx.x*(NUM_TESTS + 1)] = md->cv_nlscoef / md->cv_tq[2+blockIdx.x*(NUM_TESTS + 1)];
}

__device__
void cudaDevicecvSetBDF(ModelDataGPU *md, ModelDataVariable *sc) {
  extern __shared__ int flag_shr[];
  double alpha0, alpha0_hat, xi_inv, xistar_inv, hsum;
  int z,j;
  __syncthreads();
  md->cv_l[0+blockIdx.x*L_MAX] = md->cv_l[1+blockIdx.x*L_MAX] = xi_inv = xistar_inv = 1.;
  for (z=2; z <= sc->cv_q; z++) md->cv_l[z+blockIdx.x*L_MAX] = 0.;
  alpha0 = alpha0_hat = -1.;
  hsum = sc->cv_h;
  __syncthreads();
  if (sc->cv_q > 1) {
    for (j=2; j < sc->cv_q; j++) {
      hsum += md->cv_tau[j-1+blockIdx.x*(L_MAX + 1)];
      xi_inv = sc->cv_h / hsum;
      alpha0 -= 1. / j;
      for (z=j; z >= 1; z--) md->cv_l[z+blockIdx.x*L_MAX] += md->cv_l[z-1+blockIdx.x*L_MAX]*xi_inv;
    }
    __syncthreads();
    alpha0 -= 1. / sc->cv_q;
    xistar_inv = -md->cv_l[1+blockIdx.x*L_MAX] - alpha0;
    hsum += md->cv_tau[sc->cv_q-1+blockIdx.x*(L_MAX + 1)];
    xi_inv = sc->cv_h / hsum;
    alpha0_hat = -md->cv_l[1+blockIdx.x*L_MAX] - xi_inv;
    for (z=sc->cv_q; z >= 1; z--)
      md->cv_l[z+blockIdx.x*L_MAX] += md->cv_l[z-1+blockIdx.x*L_MAX]*xistar_inv;
  }
  __syncthreads();
  cudaDevicecvSetTqBDFt(md, sc, hsum, alpha0, alpha0_hat, xi_inv, xistar_inv);
}

__device__
void cudaDevicecvSet(ModelDataGPU *md, ModelDataVariable *sc) {
  extern __shared__ int flag_shr[];
  __syncthreads();
  cudaDevicecvSetBDF(md,sc);
  __syncthreads();
  sc->cv_rl1 = 1.0 / md->cv_l[1+blockIdx.x*L_MAX];
  sc->cv_gamma = sc->cv_h * sc->cv_rl1;
  __syncthreads();
  if (sc->cv_nst == 0){
    sc->cv_gammap = sc->cv_gamma;
  }
  __syncthreads();
  sc->cv_gamrat = (sc->cv_nst > 0) ?
                    sc->cv_gamma / sc->cv_gammap : 1.;  // protect x / x != 1.0
  __syncthreads();
}

__device__
void cudaDevicecvPredict(ModelDataGPU *md, ModelDataVariable *sc) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j, k;
  __syncthreads();
  sc->cv_tn += sc->cv_h;
  __syncthreads();
  if (md->cv_tstopset) {
    if ((sc->cv_tn - md->cv_tstop)*sc->cv_h > 0.)
      sc->cv_tn = md->cv_tstop;
  }
  md->cv_last_yn[i]=md->dzn[i];
  __syncthreads();
  print_double(md->dzn,86,"dzn1432");
  for (k = 1; k <= sc->cv_q; k++){
    for (j = sc->cv_q; j >= k; j--){
      md->dzn[i+md->nrows*(j-1)]+=md->dzn[i+md->nrows*j];
      __syncthreads();
    }
  }
  print_double(md->dzn,86,"dzn1439");
}

__device__
void cudaDevicecvDecreaseBDF(ModelDataGPU *md, ModelDataVariable *sc) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double hsum, xi;
  int z, j;
  for (z=0; z <= md->cv_qmax; z++) md->cv_l[z+blockIdx.x*L_MAX] = 0.;
  md->cv_l[2+blockIdx.x*L_MAX] = 1.;
  hsum = 0.;
  for (j=1; j <= sc->cv_q-2; j++) {
    hsum += md->cv_tau[j+blockIdx.x*(L_MAX + 1)];
    xi = hsum /sc->cv_hscale;
    for (z=j+2; z >= 2; z--)
      md->cv_l[z+blockIdx.x*L_MAX] = md->cv_l[z+blockIdx.x*L_MAX]*xi + md->cv_l[z-1+blockIdx.x*L_MAX];
  }
  for (j=2; j < sc->cv_q; j++){
    md->dzn[i+md->nrows*j]=-md->cv_l[j+blockIdx.x*L_MAX]*
      md->dzn[i+md->nrows*sc->cv_q]+md->dzn[i+md->nrows*j];
  }
  print_double(md->dzn,86,"dzn1460");
}

__device__
int cudaDevicecvDoErrorTest(ModelDataGPU *md, ModelDataVariable *sc,
       int *nflagPtr,double saved_t, int *nefPtr, double *dsmPtr) {
  extern __shared__ double flag_shr2[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double dsm;
  double min_val;
  int retval;
  md->dftemp[i]=md->cv_l[blockIdx.x*L_MAX]*md->cv_acor[i]+md->dzn[i];
  cudaDevicemin_2(&min_val, md->dftemp[i], flag_shr2, md->n_shr_empty);
  if (min_val < 0. && min_val > -CAMP_TINY) {
    md->dftemp[i]=fabs(md->dftemp[i]);
    md->dzn[i]=md->dftemp[i]-md->cv_l[0+blockIdx.x*L_MAX]*md->cv_acor[i];
    min_val = 0.;
  }
  print_double(md->dzn,86,"dzn1487");
  //print_double(&md->cv_tq[2+blockIdx.x*(NUM_TESTS + 1)],1,"cv_tq_21504");
  //print_double(&sc->cv_acnrm,1,"cv_acnrm1504");
  dsm = sc->cv_acnrm * md->cv_tq[2+blockIdx.x*(NUM_TESTS + 1)];
  //print_double(&dsm,1,"dsm1504");
  *dsmPtr = dsm;
  if (dsm <= 1. && min_val >= 0.) return(CV_SUCCESS);
  (*nefPtr)++;
  *nflagPtr = PREV_ERR_FAIL;
  cudaDevicecvRestore(md, sc, saved_t);
  __syncthreads();
  if ((fabs(sc->cv_h) <= sc->cv_hmin*ONEPSM) ||
      (*nefPtr == md->cv_maxnef)) return(CV_ERR_FAILURE);
  sc->cv_etamax = 1.;
  __syncthreads();
  if (*nefPtr <= MXNEF1) {
    sc->cv_eta = 1. / (dSUNRpowerR(BIAS2*dsm,1./sc->cv_L) + ADDON);
    __syncthreads();
    sc->cv_eta = SUNMAX(ETAMIN, SUNMAX(sc->cv_eta,
                           sc->cv_hmin / fabs(sc->cv_h)));
    __syncthreads();
    if (*nefPtr >= SMALL_NEF)
      sc->cv_eta = SUNMIN(sc->cv_eta, ETAMXF);
    __syncthreads();
    //print_double(&sc->cv_eta,1,"cv_eta_1510");
    cudaDevicecvRescale(md, sc);
    return(TRY_AGAIN);
  }
  __syncthreads();
  if (sc->cv_q > 1) {
    sc->cv_eta = SUNMAX(ETAMIN,sc->cv_hmin / fabs(sc->cv_h));
    //print_double(&sc->cv_eta,1,"cv_eta_1517");
    cudaDevicecvDecreaseBDF(md, sc);
    sc->cv_L = sc->cv_q;
    //print_int(&sc->cv_L,1,"cv_L1547");
    sc->cv_q--;
    sc->cv_qwait = sc->cv_L;
    cudaDevicecvRescale(md, sc);
    __syncthreads();
    return(TRY_AGAIN);
  }
  __syncthreads();
  sc->cv_eta = SUNMAX(ETAMIN, sc->cv_hmin / fabs(sc->cv_h));
  //print_double(&sc->cv_eta,1,"cv_eta_1529");
  __syncthreads();
  sc->cv_h *= sc->cv_eta;
  sc->cv_next_h = sc->cv_h;
  sc->cv_hscale = sc->cv_h;
  __syncthreads();
  sc->cv_qwait = 10;
  int aux_flag=0;
  retval=cudaDevicef(sc->cv_tn, md->dzn, md->dtempv,md,sc, &aux_flag);
  if (retval < 0)  return(CV_RHSFUNC_FAIL);
  if (retval > 0)  return(CV_UNREC_RHSFUNC_ERR);
  md->dzn[i+md->nrows]=sc->cv_h*md->dtempv[i];
  return(TRY_AGAIN);
}

__device__
void cudaDevicecvCompleteStep(ModelDataGPU *md, ModelDataVariable *sc) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int z, j;
  __syncthreads();
  sc->cv_nst++;
  __syncthreads();
  sc->cv_hu = sc->cv_h;
  for (z=sc->cv_q; z >= 2; z--)  md->cv_tau[z+blockIdx.x*(L_MAX + 1)] = md->cv_tau[z-1+blockIdx.x*(L_MAX + 1)];
  if ((sc->cv_q==1) && (sc->cv_nst > 1))
    md->cv_tau[2+blockIdx.x*(L_MAX + 1)] = md->cv_tau[1+blockIdx.x*(L_MAX + 1)];
  md->cv_tau[1+blockIdx.x*(L_MAX + 1)] = sc->cv_h;
  __syncthreads();
  for (j=0; j <= sc->cv_q; j++){
    md->dzn[i+md->nrows*j]+=md->cv_l[j+blockIdx.x*L_MAX]*md->cv_acor[i];
    __syncthreads();
  }
  print_double(md->dzn,86,"dzn1554");
  sc->cv_qwait--;
  if ((sc->cv_qwait == 1) && (sc->cv_q != md->cv_qmax)) {
    md->dzn[i+md->nrows*md->cv_qmax]=md->cv_acor[i];
    sc->cv_saved_tq5 = md->cv_tq[5+blockIdx.x*(NUM_TESTS + 1)];
    sc->cv_indx_acor = md->cv_qmax;
  }
}

__device__
void cudaDevicecvChooseEta(ModelDataGPU *md, ModelDataVariable *sc) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double etam;
  //print_double(&sc->cv_etaqm1,1,"cv_etaqm1605");
  //print_double(&sc->cv_etaq,1,"cv_etaq1605");
  //print_double(&sc->cv_etaqp1,1,"cv_etaqp1605");
  etam = SUNMAX(sc->cv_etaqm1, SUNMAX(sc->cv_etaq, sc->cv_etaqp1));
  //print_double(&etam,1,"etam1605");
  __syncthreads();
  if (etam < THRESH) {
    sc->cv_eta = 1.;
    //print_double(&sc->cv_eta,1,"cv_eta1609");
    sc->cv_qprime = sc->cv_q;
    return;
  }
  __syncthreads();
  if (etam == sc->cv_etaq) {
    sc->cv_eta = sc->cv_etaq;
    //print_double(&sc->cv_eta,1,"cv_eta1616");
    sc->cv_qprime = sc->cv_q;
  } else if (etam == sc->cv_etaqm1) {
    sc->cv_eta = sc->cv_etaqm1;
    //print_double(&sc->cv_eta,1,"cv_eta1620");
    sc->cv_qprime = sc->cv_q - 1;
  } else {
    sc->cv_eta = sc->cv_etaqp1;
    //print_double(&sc->cv_eta,1,"cv_eta1624");
    sc->cv_qprime = sc->cv_q + 1;
    __syncthreads();
    md->dzn[i+md->nrows*md->cv_qmax]=md->cv_acor[i];
  }
  __syncthreads();
  print_double(md->dzn,86,"dzn1581");
}

__device__
void cudaDevicecvSetEta(ModelDataGPU *md, ModelDataVariable *sc) {
  __syncthreads();
  if (sc->cv_eta < THRESH) {
    sc->cv_eta = 1.;
    sc->cv_hprime = sc->cv_h;
  } else {
    __syncthreads();
    sc->cv_eta = SUNMIN(sc->cv_eta, sc->cv_etamax);
    __syncthreads();
    sc->cv_eta /= SUNMAX(ONE,
            fabs(sc->cv_h)*md->cv_hmax_inv*sc->cv_eta);
    __syncthreads();
    sc->cv_hprime = sc->cv_h * sc->cv_eta;
    __syncthreads();
  }
  //print_double(&sc->cv_eta,1,"cv_eta_1618");
  __syncthreads();
}

__device__
int cudaDevicecvPrepareNextStep(ModelDataGPU *md, ModelDataVariable *sc, double dsm) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  __syncthreads();
  if (sc->cv_etamax == 1.) {
    sc->cv_qwait = SUNMAX(sc->cv_qwait, 2);
    sc->cv_qprime = sc->cv_q;
    sc->cv_hprime = sc->cv_h;
    sc->cv_eta = 1.;
    //print_double(&sc->cv_eta,1,"cv_eta_1631");
    return 0;
  }
  __syncthreads();
  //print_double(&dsm,1,"dsm1639");
  //print_int(&sc->cv_L,1,"cv_L1639");
  //double BIAS2dsm=BIAS2*dsm;
  //print_double(&BIAS2dsm,1,"BIAS2dsm");
  //double cv_L1=sc->cv_L;
  //print_double(&cv_L1,1,"1cv_L");
  //double cv_etaq_power=dSUNRpowerR(BIAS2dsm,cv_L1);
  //print_double(&cv_etaq_power,1,"cv_etaq_power");
  //double cv_etaq_sqrt=sqrt(BIAS2dsm);
  //print_double(&cv_etaq_sqrt,1,"cv_etaq_sqrt");
  sc->cv_etaq=1./(dSUNRpowerR(BIAS2*dsm,1./sc->cv_L) + ADDON);
  //print_double(&sc->cv_etaq,1,"cv_etaq1639");
  /*
  if(sc->cv_L!=2){
    //print_int(&sc->cv_L,1,"cv_L1674");
    if(i==0)printf("WARNING: pow is innacurate from CPU"
    " result for CUDA/10.1.105 "
    " (which is used during development at CTE-POWER) "
    " (debug by compare pow(x,0.5) and"
    " sqrt(x.0.5), double x=3.28586921557249207e-12)\n");
  }
   */
  __syncthreads();
  if (sc->cv_qwait != 0) {
    sc->cv_eta = sc->cv_etaq;
    //print_double(&sc->cv_eta,1,"cv_eta1639");
    sc->cv_qprime = sc->cv_q;
    cudaDevicecvSetEta(md, sc);
    return 0;
  }
  __syncthreads();
  sc->cv_qwait = 2;
  double ddn;
  sc->cv_etaqm1 = 0.;
  __syncthreads();
  if (sc->cv_q > 1) {
    cudaDeviceVWRMS_Norm_2(&md->dzn[md->nrows*sc->cv_q],
                         md->dewt, &ddn, md->n_shr_empty);
    __syncthreads();
    ddn *= md->cv_tq[1+blockIdx.x*(NUM_TESTS + 1)];
    __syncthreads();
    sc->cv_etaqm1 = 1./(dSUNRpowerR(BIAS1*ddn, 1./sc->cv_q) + ADDON);
  }
  double dup, cquot;
  sc->cv_etaqp1 = 0.;
  __syncthreads();
  if (sc->cv_q != md->cv_qmax && sc->cv_saved_tq5 != 0.) {
    cquot = (md->cv_tq[5+blockIdx.x*(NUM_TESTS + 1)] / sc->cv_saved_tq5) *
            dSUNRpowerI(sc->cv_h/md->cv_tau[2+blockIdx.x*(L_MAX + 1)],(double)sc->cv_L);
    md->dtempv[i]=md->cv_acor[i]-cquot*md->dzn[i+md->nrows*md->cv_qmax];
    //print_double(md->dtempv,86,"dtempv1658");
    cudaDeviceVWRMS_Norm_2(md->dtempv, md->dewt, &dup, md->n_shr_empty);
    __syncthreads();
    dup *= md->cv_tq[3+blockIdx.x*(NUM_TESTS + 1)];
    __syncthreads();
    //print_double(&dup,1,"dup1728");
    //print_int(&sc->cv_L,1,"cv_L1728");
    //double BIAS3dup=BIAS3*dup;
    //print_double(&BIAS3dup,1,"BIAS3dup");
    //double cv_L1=1./(sc->cv_L+1);
    //print_double(&cv_L1,1,"1cv_L1732");
    //double cv_etaq_power=dSUNRpowerR(BIAS3dup,1./cv_L1);
    //double cv_etaq_power=(double)pow((double)BIAS3dup,(double)cv_L1);
    //print_double(&cv_etaq_power,1,"cv_etaq_power1734");
    sc->cv_etaqp1 = 1. / (dSUNRpowerR(BIAS3*dup, 1./(sc->cv_L+1)) + ADDON);
    //print_double(&sc->cv_etaqp1,1,"cv_etaqp1728");
  }
  __syncthreads();
  cudaDevicecvChooseEta(md, sc);
  __syncthreads();
  cudaDevicecvSetEta(md, sc);
  __syncthreads();
  return CV_SUCCESS;
}

__device__
void cudaDevicecvIncreaseBDF(ModelDataGPU *md, ModelDataVariable *sc) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double alpha0, alpha1, prod, xi, xiold, hsum, A1;
  int z, j;
  for (z=0; z <= md->cv_qmax; z++) md->cv_l[z+blockIdx.x*L_MAX] = 0.;
  md->cv_l[2+blockIdx.x*L_MAX] = alpha1 = prod = xiold = 1.;
  alpha0 = -1.;
  hsum = sc->cv_hscale;
  if (sc->cv_q > 1) {
    for (j=1; j < sc->cv_q; j++) {
      hsum += md->cv_tau[j+1+blockIdx.x*(L_MAX + 1)];
      xi = hsum / sc->cv_hscale;
      prod *= xi;
      alpha0 -= 1. / (j+1);
      alpha1 += 1. / xi;
      for (z=j+2; z >= 2; z--)
        md->cv_l[z+blockIdx.x*L_MAX] = md->cv_l[z+blockIdx.x*L_MAX]*xiold + md->cv_l[z-1+blockIdx.x*L_MAX];
      xiold = xi;
    }
  }
  A1 = (-alpha0 - alpha1) / prod;
  md->dzn[i+md->nrows*sc->cv_L]=A1*md->dzn[i+md->nrows*sc->cv_indx_acor];
  __syncthreads();
  for (j=2; j <= sc->cv_q; j++){
    md->dzn[i+md->nrows*j]+=md->cv_l[j+blockIdx.x*L_MAX]*md->dzn[i+md->nrows*(sc->cv_L)];
    __syncthreads();
  }
  print_double(md->dzn,86,"dzn1687");
}

__device__
void cudaDevicecvAdjustParams(ModelDataGPU *md, ModelDataVariable *sc) {
  if (sc->cv_qprime != sc->cv_q) {
    int deltaq = sc->cv_qprime-sc->cv_q;
    switch(deltaq) {
      case 1:
        cudaDevicecvIncreaseBDF(md, sc);
        break;
      case -1:
        cudaDevicecvDecreaseBDF(md, sc);
        break;
    }
    sc->cv_q = sc->cv_qprime;
    sc->cv_L = sc->cv_q+1;
    //print_int(&sc->cv_L,1,"cv_L1770");
    sc->cv_qwait = sc->cv_L;
  }
  cudaDevicecvRescale(md, sc);
}

__device__
int cudaDevicecvStep(ModelDataGPU *md, ModelDataVariable *sc) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int ncf = 0;
  int nef = 0;
  int nflag=FIRST_CALL;
  double saved_t=sc->cv_tn;
  double dsm;
  __syncthreads();
  if ((sc->cv_nst > 0) && (sc->cv_hprime != sc->cv_h)){
    cudaDevicecvAdjustParams(md, sc);
  }
  __syncthreads();
  for (;;) {
    __syncthreads();
    cudaDevicecvPredict(md, sc);
    __syncthreads();
    cudaDevicecvSet(md, sc);
    __syncthreads();
    nflag = cudaDevicecvNlsNewton(nflag,md, sc);
    __syncthreads();
    int kflag = cudaDevicecvHandleNFlag(md, sc, &nflag, saved_t, &ncf);
    __syncthreads();
    if (kflag == PREDICT_AGAIN) {
      continue;
    }
    if (kflag != DO_ERROR_TEST) {
      return (kflag);
    }
    __syncthreads();
    int eflag=cudaDevicecvDoErrorTest(md,sc,&nflag,saved_t,&nef,&dsm);
    __syncthreads();
    if (eflag == TRY_AGAIN){
      continue;
    }
    if (eflag != CV_SUCCESS){
      return (eflag);
    }
    break;
  }
  __syncthreads();
  cudaDevicecvCompleteStep(md, sc);
  __syncthreads();
  cudaDevicecvPrepareNextStep(md, sc, dsm);
  __syncthreads();
  sc->cv_etamax=10.;
  md->cv_acor[i]*=md->cv_tq[2+blockIdx.x*(NUM_TESTS + 1)];
  __syncthreads();
  return(CV_SUCCESS);
  }

__device__
int cudaDeviceCVodeGetDky(ModelDataGPU *md, ModelDataVariable *sc,
                           double t, int k, double *dky) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double s, c, r;
  double tfuzz, tp, tn1;
  int z, j;
  __syncthreads();
   tfuzz = FUZZ_FACTOR * md->cv_uround * (fabs(sc->cv_tn) + fabs(sc->cv_hu));
   if (sc->cv_hu < 0.) tfuzz = -tfuzz;
   tp = sc->cv_tn - sc->cv_hu - tfuzz;
   tn1 = sc->cv_tn + tfuzz;
   if ((t-tp)*(t-tn1) > 0.) {
     return(CV_BAD_T);
   }
  __syncthreads();
   s = (t - sc->cv_tn) / sc->cv_h;
   for (j=sc->cv_q; j >= k; j--) {
     c = 1.;
     for (z=j; z >= j-k+1; z--) c *= z;
     if (j == sc->cv_q) {
       dky[i]=c*md->dzn[i+md->nrows*j];
     } else {
        dky[i]=c*md->dzn[i+md->nrows*j]+s*dky[i];
     }
   }
  __syncthreads();
   if (k == 0) return(CV_SUCCESS);
  __syncthreads();
   r = dSUNRpowerI(double(sc->cv_h),double(-k));
  __syncthreads();
   dky[i]=dky[i]*r;
   return(CV_SUCCESS);
}

__device__
int cudaDevicecvEwtSetSV(ModelDataGPU *md, ModelDataVariable *sc,double *weight) {
  extern __shared__ double flag_shr2[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  md->dtempv[i]=fabs(md->dzn[i]);
  double min;
  md->dtempv[i]=md->cv_reltol*md->dtempv[i]+md->cv_Vabstol[i];
  cudaDevicemin_2(&min, md->dtempv[i], flag_shr2, md->n_shr_empty);
__syncthreads();
  if (min <= 0.) return(-1);
  weight[i]= 1./md->dtempv[i];
  return(0);
}

__device__
int cudaDeviceCVode(ModelDataGPU *md, ModelDataVariable *sc) {
  extern __shared__ int flag_shr[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int kflag2;
  sc->nstloc=0;
  sc->nstlj=0;
  sc->cv_nst=0;
  sc->cv_nstlp=0;
  for(;;) {
    __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0) sc->countercvStep++;
#endif
    flag_shr[0] = 0;
    __syncthreads();
    sc->cv_next_h = sc->cv_h;
    int ewtsetOK = 0;
    if (sc->cv_nst > 0) {
      //print_double(md->dtempv,86,"dtempvcv_efun0");
      ewtsetOK = cudaDevicecvEwtSetSV(md, sc, md->dewt);
      //print_double(md->dtempv,86,"dtempvcv_efun1");
      if (ewtsetOK != 0) {
        sc->cv_tretlast = sc->tret = sc->cv_tn;
        md->yout[i] = md->dzn[i];
        if(i==0) printf("ERROR: ewtsetOK\n");
        return CV_ILL_INPUT;
      }
    }
    if ((md->cv_mxstep > 0) && (sc->nstloc >= md->cv_mxstep)) {
      sc->cv_tretlast = sc->tret = sc->cv_tn;
      md->yout[i] = md->dzn[i];
      if(i==0) printf("ERROR: cv_mxstep reached "
        "sc->nstloc %d md->cv_mxstep %d\n",sc->nstloc,md->cv_mxstep);
      return CV_TOO_MUCH_WORK;
    }
    double nrm;
    cudaDeviceVWRMS_Norm_2(md->dzn,
     md->dewt, &nrm, md->n_shr_empty);
    sc->cv_tolsf = md->cv_uround * nrm;
    if (sc->cv_tolsf > 1.) {
      sc->cv_tretlast = sc->tret = sc->cv_tn;
      md->yout[i] = md->dzn[i];
      sc->cv_tolsf *= 2.;
      if(i==0) printf("ERROR: cv_tolsf\n");
      __syncthreads();
      if(i==0) printf("ERROR: cv_tolsf\n");
      return CV_TOO_MUCH_ACC;
    } else {
      sc->cv_tolsf = 1.;
    }
#ifdef ODE_WARNING
    if (sc->cv_tn + sc->cv_h == sc->cv_tn) {
      if(threadIdx.x==0) sc->cv_nhnil++;
      if ((sc->cv_nhnil <= sc->cv_mxhnil) ||
              (sc->cv_nhnil == sc->cv_mxhnil))
        if(i==0)printf("WARNING: h below roundoff level in tn");
    }
#endif
    print_double(md->dzn,86,"dzn1858");
    kflag2 = cudaDevicecvStep(md, sc);
    __syncthreads();
    if (kflag2 != CV_SUCCESS) {
      sc->cv_tretlast = sc->tret = sc->cv_tn;
      md->yout[i] = md->dzn[i];
      if(i==0) printf("ERROR: kflag != CV_SUCCESS\n");
      return kflag2;
    }
    sc->nstloc++;
    if ((sc->cv_tn - md->tout) * sc->cv_h >= 0.) {
      sc->cv_tretlast = sc->tret = md->tout;
      cudaDeviceCVodeGetDky(md, sc, md->tout, 0, md->yout);
      return CV_SUCCESS;
    }
    if (md->cv_tstopset) {//needed?
      double troundoff = FUZZ_FACTOR * md->cv_uround * (fabs(sc->cv_tn) + fabs(sc->cv_h));
      if (fabs(sc->cv_tn - md->cv_tstop) <= troundoff) {
        cudaDeviceCVodeGetDky(md, sc, md->cv_tstop, 0, md->yout);
        sc->cv_tretlast = sc->tret = md->cv_tstop;
        md->cv_tstopset = SUNFALSE;
        if(i==0) printf("ERROR: cv_tstopset\n");
        __syncthreads();
        return CV_TSTOP_RETURN;
      }
      if ((sc->cv_tn + sc->cv_hprime - md->cv_tstop) * sc->cv_h > 0.) {
        sc->cv_hprime = (md->cv_tstop - sc->cv_tn) * (1.0 - 4.0 * md->cv_uround);
        if(i==0) printf("ERROR: sc->cv_tn + sc->cv_hprime - sc->cv_tstop\n");
        sc->cv_eta = sc->cv_hprime / sc->cv_h;
      }
    }
  }
}

__global__
void cudaGlobalCVode(ModelDataGPU md_object) {
  ModelDataGPU *md = &md_object;
  extern __shared__ int flag_shr[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  //TODO CHECK IF USING SC AS LOCAL INSTEAD OF MD->SCELLS HAS BETTER MAPE AND FINE IN MONARCH
  //IF WANT TO USE SC 1 PER BLOCK, THEN CHECK ALL SC->SOMETHING = SOMETHING AND BLOCKIDX.X CALLS AND ADD IF(THREADIDX.X==0)...SYNCTHREADS() TO AVOID OVERLAPPING
  //ModelDataVariable *sc = &md->sCells[blockIdx.x];
  ModelDataVariable sc_object = md->sCells[blockIdx.x];
  ModelDataVariable *sc = &sc_object;
  __syncthreads();
  int istate;
  if(i<md->nrows){
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    int clock_khz=md->clock_khz;
    clock_t start;
    start = clock();
    __syncthreads();
#endif
    istate=cudaDeviceCVode(md,sc);
    __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  if(threadIdx.x==0) sc->dtcudaDeviceCVode += ((double)(int)(clock() - start))/(clock_khz*1000);
  __syncthreads();
#endif
  }
  __syncthreads();
  if(threadIdx.x==0) md->flagCells[blockIdx.x]=istate;
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  ModelDataVariable *mdvo = md->mdvo;
  *mdvo = *sc;
#endif
}

int nextPowerOfTwoCVODE2(int v){
  v--;
  v |= v >> 1;
  v |= v >> 2;
  v |= v >> 4;
  v |= v >> 8;
  v |= v >> 16;
  v++;
  return v;
}

void cvodeRun(ModelDataGPU *mGPU, hipStream_t stream){
  int len_cell = mGPU->nrows / mGPU->n_cells;
  int threads_block = len_cell;
  int blocks = mGPU->n_cells;
  int n_shr_memory = nextPowerOfTwoCVODE2(len_cell);
  mGPU->n_shr_empty = n_shr_memory - threads_block;
  cudaGlobalCVode <<<blocks, threads_block, n_shr_memory * sizeof(double), stream>>>(*mGPU);
}
