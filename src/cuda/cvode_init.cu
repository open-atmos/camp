/* Copyright (C) 2021 Barcelona Supercomputing Center and University of
 * Illinois at Urbana-Champaign
 * SPDX-License-Identifier: MIT
 */

#include "cvode_cuda.h"
extern "C" {
#include "cvode_gpu.h"
}
#ifdef CAMP_USE_MPI
#include <mpi.h>
#endif

void constructor_cvode_gpu(SolverData *sd){
  CVodeMem cv_mem = (CVodeMem) sd->cvode_mem;
  ModelDataCPU *mCPU = &(sd->mCPU);
  ModelData *md = &(sd->model_data);
  CVDlsMem cvdls_mem = (CVDlsMem) cv_mem->cv_lmem;
  SUNMatrix J = cvdls_mem->A;
  sd->mGPU = (ModelDataGPU *)malloc(sizeof(ModelDataGPU));
  ModelDataGPU *mGPU = sd->mGPU;
#ifdef DEV_CPU_GPU
  int n_cells=md->n_cells_gpu; //todo use only mgpu->n_cells
#else
  int n_cells = md->n_cells;
#endif
  mGPU->n_cells= n_cells;
  sd->flagCells = (int *) malloc((n_cells) * sizeof(int));
  int n_dep_var = md->n_per_cell_dep_var;
  int n_state_var = md->n_per_cell_state_var;
  int n_rxn = md->n_rxn;
  size_t state_size = n_state_var * n_cells * sizeof(double);
  mCPU->deriv_size = n_dep_var * n_cells * sizeof(double);
  mCPU->env_size = CAMP_NUM_ENV_PARAM_ * n_cells * sizeof(double); //Temp and pressure
  size_t rxn_env_data_idx_size = (n_rxn+1) * sizeof(int);
  size_t map_state_deriv_size = n_dep_var * n_cells * sizeof(int);
  int coresPerNode = 40;
  int size;
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  if (size > 40 && size % coresPerNode != 0) {
    printf("ERROR: MORE THAN 40 MPI PROCESSES AND NOT MULTIPLE OF 40, WHEN CTE-POWER ONLY HAS 40 CORES PER NODE\n");
    exit(0);
  }
  int nGPUsMax=4;
  hipGetDeviceCount(&nGPUsMax);
  if (sd->nGPUs > nGPUsMax) {
    printf("ERROR: Not enough GPUs to launch, nGPUs %d nGPUsMax %d\n", sd->nGPUs, nGPUsMax);
    exit(0);
  }
  if (size > sd->nGPUs*(coresPerNode/nGPUsMax)){
    printf("ERROR: size,sd->nGPUs,coresPerNode,nGPUsMax %d %d %d %d "
           "MORE MPI PROCESSES THAN DEVICES (FOLLOW PROPORTION, "
           "FOR CTE-POWER IS 10 PROCESSES FOR EACH GPU)\n",size,sd->nGPUs,coresPerNode,nGPUsMax);
    exit(0);
  }
  int rank;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  hipSetDevice(0);
  for (int i = 0; i < coresPerNode; i++) {
    if (rank < coresPerNode / nGPUsMax * (i + 1) && rank >= coresPerNode / nGPUsMax * i && i<sd->nGPUs) {
      hipSetDevice(i);
      mCPU->threads = 1024;
      mCPU->blocks = (n_dep_var*n_cells + mCPU->threads - 1) / mCPU->threads;
    }
  }
  mGPU->n_rxn=md->n_rxn;
  mGPU->n_rxn_env_data=md->n_rxn_env_data;
  hipMalloc((void **) &mGPU->state, state_size);
  hipMalloc((void **) &mGPU->env, mCPU->env_size);
  hipMalloc((void **) &mGPU->rxn_env_data, md->n_rxn_env_data * n_cells * sizeof(double));
  hipMalloc((void **) &mGPU->rxn_env_data_idx, rxn_env_data_idx_size);
  hipMalloc((void **) &mGPU->map_state_deriv, map_state_deriv_size);
  int num_spec = md->n_per_cell_dep_var*n_cells;
  hipMalloc((void **) &(mGPU->production_rates),num_spec*sizeof(mGPU->production_rates));
  hipMalloc((void **) &(mGPU->loss_rates),num_spec*sizeof(mGPU->loss_rates));
  int *map_state_derivCPU = (int *)malloc(map_state_deriv_size);
  int i_dep_var = 0;
  for (int i_cell = 0; i_cell < n_cells; i_cell++) {
    for (int i_spec = 0; i_spec < n_state_var; i_spec++) {
      if (md->var_type[i_spec] == CHEM_SPEC_VARIABLE) {
        map_state_derivCPU[i_dep_var] = i_spec + i_cell * n_state_var;
        i_dep_var++;
      }
    }
  }
  HANDLE_ERROR(hipMemcpy(mGPU->map_state_deriv, map_state_derivCPU,
                          map_state_deriv_size, hipMemcpyHostToDevice));
  free(map_state_derivCPU);
  if(n_dep_var<32) {
    printf("CAMP ERROR: TOO FEW SPECIES FOR GPU (Species < 32),"
           " use CPU case instead\n");
    exit(0);
}
  mCPU->jac_size = md->n_per_cell_solver_jac_elem * n_cells * sizeof(double);
  mCPU->nnz_J_solver = SM_NNZ_S(md->J_solver);
  hipMalloc((void **) &mGPU->dA, mCPU->jac_size);
  hipMalloc((void **) &mGPU->J_solver, mCPU->jac_size);
  hipMalloc((void **) &mGPU->J_state, mCPU->deriv_size);
  hipMalloc((void **) &mGPU->J_deriv, mCPU->deriv_size);
  hipMalloc((void **) &mGPU->J_tmp, mCPU->deriv_size);
  hipMalloc((void **) &mGPU->J_tmp2, mCPU->deriv_size);
  hipMalloc((void **) &mGPU->jac_map, sizeof(JacMap) * md->n_mapped_values);
  HANDLE_ERROR(hipMalloc((void **) &mGPU->n_mapped_values, 1 * sizeof(int)));
  mCPU->A = ((double *) SM_DATA_S(J));
  HANDLE_ERROR(hipMemcpy(mGPU->dA, mCPU->A, mCPU->jac_size, hipMemcpyHostToDevice));
  double *J_solver = SM_DATA_S(md->J_solver);
  hipMemcpy(mGPU->J_solver, J_solver, mCPU->jac_size, hipMemcpyHostToDevice);
  double *J_state = N_VGetArrayPointer(md->J_state);
  HANDLE_ERROR(hipMemcpy(mGPU->J_state, J_state, mCPU->deriv_size, hipMemcpyHostToDevice));
  double *J_deriv = N_VGetArrayPointer(md->J_deriv);
  HANDLE_ERROR(hipMemcpy(mGPU->J_deriv, J_deriv, mCPU->deriv_size, hipMemcpyHostToDevice));
  double *J_tmp2 = N_VGetArrayPointer(md->J_tmp2);
  HANDLE_ERROR(hipMemcpy(mGPU->J_tmp2, J_tmp2, mCPU->deriv_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->jac_map, md->jac_map, sizeof(JacMap) * md->n_mapped_values, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->n_mapped_values, &md->n_mapped_values, 1 * sizeof(int), hipMemcpyHostToDevice));
  Jacobian *jac = &sd->jac;
  JacobianGPU *jacgpu = &(mGPU->jac);
  hipMalloc((void **) &jacgpu->num_elem, 1 * sizeof(jacgpu->num_elem));
  hipMemcpy(jacgpu->num_elem, &jac->num_elem, 1 * sizeof(jacgpu->num_elem), hipMemcpyHostToDevice);
  int num_elem = jac->num_elem * n_cells;
  hipMalloc((void **) &(jacgpu->production_partials), num_elem * sizeof(double));
  HANDLE_ERROR(hipMalloc((void **) &(jacgpu->loss_partials), num_elem * sizeof(double)));
  double *aux=(double*)malloc(sizeof(double)*num_elem);
  for (int i = 0; i < num_elem; i++) {
    aux[i]=0.;
  }
  HANDLE_ERROR(hipMemcpy(jacgpu->production_partials, aux, num_elem * sizeof(double), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(jacgpu->loss_partials, aux, num_elem * sizeof(double), hipMemcpyHostToDevice));
  hipMalloc((void **) &mGPU->rxn_int, (md->n_rxn_int_param + md->n_rxn)*sizeof(int));
  hipMalloc((void **) &mGPU->rxn_double, md->n_rxn_float_param*sizeof(double));
  hipMalloc((void **) &mGPU->rxn_int_indices, (md->n_rxn+1)*sizeof(int));
  hipMalloc((void **) &mGPU->rxn_float_indices, (md->n_rxn+1)*sizeof(int));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_int, md->rxn_int_data,(md->n_rxn_int_param + md->n_rxn)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_double, md->rxn_float_data, md->n_rxn_float_param*sizeof(double), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_env_data_idx, md->rxn_env_idx, rxn_env_data_idx_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_int_indices, md->rxn_int_indices,(md->n_rxn+1)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_float_indices, md->rxn_float_indices,(md->n_rxn+1)*sizeof(int), hipMemcpyHostToDevice));
  mCPU->nnz = SM_NNZ_S(J);
  int nrows = SM_NP_S(J);
  mGPU->nrows = nrows;
  double ** dr0 = &mGPU->dr0;
  double ** dr0h = &mGPU->dr0h;
  double ** dn0 = &mGPU->dn0;
  double ** dp0 = &mGPU->dp0;
  double ** dt = &mGPU->dt;
  double ** ds = &mGPU->ds;
  double ** dy = &mGPU->dy;
  double ** ddiag = &mGPU->ddiag;
  hipMalloc(dr0,nrows*sizeof(double));
  hipMalloc(dr0h,nrows*sizeof(double));
  hipMalloc(dn0,nrows*sizeof(double));
  hipMalloc(dp0,nrows*sizeof(double));
  hipMalloc(dt,nrows*sizeof(double));
  hipMalloc(ds,nrows*sizeof(double));
  hipMalloc(dy,nrows*sizeof(double));
  HANDLE_ERROR(hipMalloc(ddiag,nrows*sizeof(double)));;
  //Translate from int64 (sunindextype) to int
  mCPU->jA = (int *) malloc(sizeof(int) *mCPU->nnz/n_cells);
  mCPU->iA = (int *) malloc(sizeof(int) * (nrows/n_cells + 1));
  for (int i = 0; i < mCPU->nnz/n_cells; i++)
    mCPU->jA[i] = SM_INDEXVALS_S(J)[i];
  for (int i = 0; i <= nrows/n_cells; i++)
    mCPU->iA[i] = SM_INDEXPTRS_S(J)[i];
  hipMalloc((void **) &mGPU->djA, mCPU->nnz/n_cells * sizeof(int));
  hipMalloc((void **) &mGPU->diA, (nrows/n_cells + 1) * sizeof(int));
  hipMemcpy(mGPU->djA, mCPU->jA, mCPU->nnz/n_cells * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->diA, mCPU->iA, (nrows/n_cells + 1) * sizeof(int), hipMemcpyHostToDevice);
  double *ewt = N_VGetArrayPointer(cv_mem->cv_ewt);
  double *tempv = N_VGetArrayPointer(cv_mem->cv_tempv);
  double *cv_last_yn = N_VGetArrayPointer(cv_mem->cv_last_yn);
  double *cv_acor_init = N_VGetArrayPointer(cv_mem->cv_acor_init);
  hipMalloc((void **) &mGPU->dftemp, mCPU->deriv_size);
  hipMalloc((void **) &mGPU->sCells, sizeof(ModelDataVariable)*n_cells);
  hipMalloc((void **) &mGPU->flag, 1 * sizeof(int));
  hipMalloc((void **) &mGPU->flagCells, n_cells * sizeof(int));
  hipMalloc((void **) &mGPU->dsavedJ, mCPU->nnz * sizeof(double));
  hipMalloc((void **) &mGPU->dewt, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->cv_acor, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->dtempv, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->dtempv1, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->dtempv2, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->dzn, nrows * (cv_mem->cv_qmax + 1) * sizeof(double));
  hipMalloc((void **) &mGPU->dcv_y, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->dx, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->cv_last_yn, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->cv_acor_init, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->cv_acor, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->yout, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->cv_l, L_MAX * n_cells * sizeof(double));
  hipMalloc((void **) &mGPU->cv_tau, (L_MAX + 1) * n_cells * sizeof(double));
  hipMalloc((void **) &mGPU->cv_tq, (NUM_TESTS + 1) * n_cells * sizeof(double));
  hipMalloc((void **) &mGPU->cv_Vabstol, nrows * sizeof(double));
  HANDLE_ERROR(hipMemset(mGPU->flagCells, CV_SUCCESS, n_cells * sizeof(int)));
  hipMemcpy(mGPU->dsavedJ, mCPU->A, mCPU->nnz * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->dewt, ewt, nrows * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->cv_acor, ewt, nrows * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->dftemp, ewt, nrows * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->dx, tempv, nrows * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->cv_last_yn, cv_last_yn, nrows * sizeof(double), hipMemcpyHostToDevice);
  HANDLE_ERROR(hipMemcpy(mGPU->cv_acor_init, cv_acor_init, nrows * sizeof(double), hipMemcpyHostToDevice));
  mGPU->state_size_cell = md->n_per_cell_state_var;
  int flag = 999;
  hipMemcpy(mGPU->flag, &flag, 1 * sizeof(int), hipMemcpyHostToDevice);
  mCPU->mdvCPU.nstlj = 0;
#ifdef CAMP_DEBUG_GPU
  hipEventCreate(&mCPU->startcvStep);
  hipEventCreate(&mCPU->stopcvStep);
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  hipMalloc((void **) &mGPU->mdvo, sizeof(ModelDataVariable));
  hipDeviceGetAttribute(&mGPU->clock_khz, hipDeviceAttributeClockRate, 0);
  mCPU->mdvCPU.countercvStep=0;
  mCPU->mdvCPU.counterBCGInternal=0;
  mCPU->mdvCPU.counterBCG=0;
  mCPU->mdvCPU.timeNewtonIteration=0.;
  mCPU->mdvCPU.timeJac=0.;
  mCPU->mdvCPU.timelinsolsetup=0.;
  mCPU->mdvCPU.timecalc_Jac=0.;
  mCPU->mdvCPU.timef=0.;
  mCPU->mdvCPU.timeguess_helper=0.;
  mCPU->mdvCPU.dtBCG=0.;
  mCPU->mdvCPU.dtcudaDeviceCVode=0.;
  mCPU->mdvCPU.dtPostBCG=0.;
  HANDLE_ERROR(hipMemcpy(mGPU->mdvo, &mCPU->mdvCPU, sizeof(ModelDataVariable), hipMemcpyHostToDevice));
#endif
#endif
  for (int i = 0; i < n_cells; i++){
    hipMemcpy(&mGPU->sCells[i], &mCPU->mdvCPU, sizeof(ModelDataVariable), hipMemcpyHostToDevice);
  }
#ifdef IS_DEBUG_MODE_CSR_ODE_GPU
  int n_row=nrows/n_cells;
  int* Ap=mCPU->iA;
  int* Aj=mCPU->jA;
  double* Ax=mCPU->A;
  int nnz=mCPU->nnz/n_cells;
  int* Bp=(int*)malloc((n_row+1)*sizeof(int));
  int* Bi=(int*)malloc(nnz*sizeof(int));
  double* Bx=(double*)malloc(nnz*sizeof(double));
  memset(Bp, 0, (n_row+1)*sizeof(int));
  for (int n = 0; n < nnz; n++){
   Bp[Aj[n]]++;
  }
  for(int col = 0, cumsum = 0; col < n_row; col++){
    int temp  = Bp[col];
    Bp[col] = cumsum;
    cumsum += temp;
  }
  Bp[n_row] = nnz;
  int *mapJSPMV= (int *)malloc(nnz * sizeof(int));
  for(int row = 0; row < n_row; row++){
    for(int jj = Ap[row]; jj < Ap[row+1]; jj++){
      int col  = Aj[jj];
      int dest = Bp[col];
      Bi[dest] = row;
      Bx[dest] = Ax[jj];
      mapJSPMV[jj]=dest;
      Bp[col]++;
    }
  }
  for(int col = 0, last = 0; col <= n_row; col++){
    int temp  = Bp[col];
    Bp[col] = last;
    last    = temp;
  }
  ModelData *md = &(sd->model_data);
  nnz=md->n_mapped_values;
  int *aux_solver_id= (int *)malloc(nnz * sizeof(int));
  for (int i = 0; i < nnz; i++){
    aux_solver_id[i]=mapJSPMV[md->jac_map[i].solver_id];
  }
  free(mapJSPMV);
  int *jac_solver_id= (int *)malloc(nnz * sizeof(int));
  for (int i = 0; i < nnz; i++){
    jac_solver_id[i]=aux_solver_id[i];
    aux_solver_id[i]=md->jac_map[i].solver_id;
    md->jac_map[i].solver_id=jac_solver_id[i];
  }
  hipMemcpy(mGPU->diA, Bp, (n_row + 1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->djA, Bi, nnz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->dA, Bx, nnz * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->jac_map, md->jac_map, sizeof(JacMap) * md->n_mapped_values, hipMemcpyHostToDevice);
  free(Bp);
  free(Bi);
  free(Bx);
  free(jac_solver_id);
  free(aux_solver_id);
#endif
}

void free_gpu_cu(SolverData *sd) {
  ModelDataGPU *mGPU = sd->mGPU;
  free(sd->flagCells);
  mGPU = sd->mGPU;
  hipFree(mGPU->map_state_deriv);
  hipFree(mGPU->J_solver);
  hipFree(mGPU->J_state);
  hipFree(mGPU->J_deriv);
  hipFree(mGPU->J_tmp);
  hipFree(mGPU->J_tmp2);
  hipFree(mGPU->indexvals);
  hipFree(mGPU->indexptrs);
  hipFree(mGPU->rxn_int);
  hipFree(mGPU->rxn_double);
  hipFree(mGPU->state);
  hipFree(mGPU->env);
  hipFree(mGPU->rxn_env_data);
  hipFree(mGPU->rxn_env_data_idx);
  hipFree(mGPU->production_rates);
  hipFree(mGPU->loss_rates);
  hipFree(mGPU->rxn_int_indices);
  hipFree(mGPU->rxn_float_indices);
  hipFree(mGPU->n_mapped_values);
  hipFree(mGPU->jac_map);
  hipFree(mGPU->yout);
  hipFree(mGPU->cv_Vabstol);
  hipFree(mGPU->cv_l);
  hipFree(mGPU->cv_tau);
  hipFree(mGPU->cv_tq);
  hipFree(mGPU->cv_last_yn);
  hipFree(mGPU->cv_acor_init);
  hipFree(mGPU->dA);
  hipFree(mGPU->djA);
  hipFree(mGPU->diA);
  hipFree(mGPU->dx);
  hipFree(mGPU->dtempv);
  hipFree(mGPU->ddiag);
  hipFree(mGPU->dr0);
  hipFree(mGPU->dr0h);
  hipFree(mGPU->dn0);
  hipFree(mGPU->dp0);
  hipFree(mGPU->dt);
  hipFree(mGPU->ds);
  hipFree(mGPU->dy);
  hipFree(mGPU->dz);
  hipFree(mGPU->dftemp);
  hipFree(mGPU->dcv_y);
  hipFree(mGPU->dtempv1);
  hipFree(mGPU->dtempv2);
  hipFree(mGPU->flag);
  hipFree(mGPU->flagCells);
  hipFree(mGPU->cv_acor);
  hipFree(mGPU->dzn);
  hipFree(mGPU->dewt);
  hipFree(mGPU->dsavedJ);
  hipFree(mGPU->mdv);
  hipFree(mGPU->mdvo);
}