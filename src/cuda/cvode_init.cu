/* Copyright (C) 2021 Barcelona Supercomputing Center and University of
 * Illinois at Urbana-Champaign
 * SPDX-License-Identifier: MIT
 */

#include "cvode_cuda.h"
extern "C" {
#include "cvode_gpu.h"
}
#ifdef CAMP_USE_MPI
#include <mpi.h>
#endif

void init_solve_gpu(SolverData *sd) {
  ModelData *md = &(sd->model_data);
  int n_dep_var = md->n_per_cell_dep_var;
#ifndef SANITY_CHECK
  if (n_dep_var < 32) {
    printf("CAMP ERROR: TOO FEW SPECIES FOR GPU, use CPU case instead\n");
    exit(0);
  }
  if (n_dep_var > 1024) {
    printf("CAMP ERROR: TOO MUCH SPECIES FOR GPU,use CPU case instead\n");
    exit(0);
  }
  // Check if reaction types are implemented in the GPU
  int n_rxn = md->n_rxn;
  for (int i_rxn = 0; i_rxn < n_rxn; i_rxn++) {
    int *rxn_int_data = &(md->rxn_int_data[md->rxn_int_indices[i_rxn]]);
    int rxn_type = rxn_int_data[0];
    switch (rxn_type) {
      case RXN_ARRHENIUS:
        break;
      case RXN_CMAQ_H2O2:
        break;
      case RXN_CMAQ_OH_HNO3:
        break;
      case RXN_FIRST_ORDER_LOSS:
        break;
      case RXN_PHOTOLYSIS:
        break;
      case RXN_TROE:
        break;
      default:
        printf("CAMP ERROR: Reaction type not implemented in GPU\n");
        exit(0);
    }
  }
#endif
  CVodeMem cv_mem = (CVodeMem)sd->cvode_mem;
  ModelDataCPU *mCPU = &(sd->mCPU);
  sd->mGPU = (ModelDataGPU *)malloc(sizeof(ModelDataGPU));
  ModelDataGPU *mGPU = sd->mGPU;
  md->n_cells_gpu = md->n_cells * sd->load_gpu / 100.;
  int n_cells = md->n_cells;  // Load balance can differ up to n_cells size
  int nrows = n_dep_var * n_cells;
  int n_state_var = md->n_per_cell_state_var;
  mGPU->n_per_cell_state_var = md->n_per_cell_state_var;
  sd->last_load_balance = 0;
  sd->last_load_gpu = 100;
  sd->acc_load_balance = 0;
  sd->iters_load_balance = 0;
  sd->last_short_gpu = 0;
  int nGPUs;
  HANDLE_ERROR(hipGetDeviceCount(&nGPUs));
  int rank;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  if (rank == 0) {
    printf("Cells to GPU: %.lf%%\n", sd->load_gpu);
  }
  int iDevice = rank % nGPUs;
  double startTime = MPI_Wtime();
  hipSetDevice(iDevice);
  if (rank == 0) printf("Time INIT: %f\n", MPI_Wtime() - startTime);
  mGPU->n_rxn = md->n_rxn;
  mGPU->n_rxn_env_data = md->n_rxn_env_data;
  HANDLE_ERROR(hipMalloc((void **)&mGPU->state,
                          n_state_var * n_cells * sizeof(double)));
  hipMalloc((void **)&mGPU->rxn_env_data,
             md->n_rxn_env_data * n_cells * sizeof(double));
  int num_spec = n_dep_var * n_cells;
  hipMalloc((void **)&(mGPU->production_rates),
             num_spec * sizeof(mGPU->production_rates));
  hipMalloc((void **)&(mGPU->loss_rates), num_spec * sizeof(mGPU->loss_rates));
  hipMalloc((void **)&mGPU->map_state_deriv, n_dep_var * sizeof(int));
  int *map_state_derivCPU = (int *)malloc(n_dep_var * sizeof(int));
  int i_dep_var = 0;
  for (int i_spec = 0; i_spec < n_state_var; i_spec++) {
    if (md->var_type[i_spec] == CHEM_SPEC_VARIABLE) {
      map_state_derivCPU[i_dep_var] = i_spec;
      i_dep_var++;
    }
  }
#ifdef PROFILE_SOLVING
  hipEventCreate(&sd->startGPU);
  hipEventCreate(&sd->stopGPU);
  hipEventCreate(&sd->startGPUSync);
  hipEventCreate(&sd->stopGPUSync);
#endif
  hipStream_t stream;
  hipStreamCreate(&stream);
  hipMemcpyAsync(mGPU->map_state_deriv, map_state_derivCPU,
                  n_dep_var * sizeof(int), hipMemcpyHostToDevice, stream);
  free(map_state_derivCPU);
  size_t deriv_size = n_dep_var * n_cells * sizeof(double);
  int nnz = md->n_per_cell_solver_jac_elem * n_cells;
  size_t jac_size = nnz * sizeof(double);
  hipMalloc((void **)&mGPU->dA, jac_size);
  hipMalloc((void **)&mGPU->J_solver, jac_size);
  hipMalloc((void **)&mGPU->J_state, deriv_size);
  double *J_state = N_VGetArrayPointer(md->J_state);
  hipMemset(mGPU->J_state, 0, deriv_size);
  hipMalloc((void **)&mGPU->J_deriv, deriv_size);
  double *J_deriv = N_VGetArrayPointer(md->J_deriv);
  hipMemset(mGPU->J_deriv, 0, deriv_size);
  hipMemset(mGPU->J_solver, 0, jac_size);
  hipMalloc((void **)&mGPU->jac_map,
             sizeof(JacMap) * md->n_per_cell_solver_jac_elem);
  hipMemcpyAsync(mGPU->jac_map, md->jac_map,
                  sizeof(JacMap) * md->n_per_cell_solver_jac_elem,
                  hipMemcpyHostToDevice, stream);
  Jacobian *jac = &sd->jac;
  JacobianGPU *jacgpu = &(mGPU->jac);
  hipMalloc((void **)&jacgpu->num_elem, 1 * sizeof(jacgpu->num_elem));
  hipMemcpyAsync(jacgpu->num_elem, &jac->num_elem,
                  1 * sizeof(jacgpu->num_elem), hipMemcpyHostToDevice, stream);
  int num_elem = jac->num_elem * n_cells;
  hipMalloc((void **)&(jacgpu->production_partials),
             num_elem * sizeof(double));
  hipMalloc((void **)&(jacgpu->loss_partials), num_elem * sizeof(double));
  hipMemset(jacgpu->production_partials, 0, num_elem * sizeof(double));
  hipMemset(jacgpu->loss_partials, 0, num_elem * sizeof(double));
  hipMalloc((void **)&mGPU->rxn_int,
             (md->n_rxn_int_param + md->n_rxn) * sizeof(int));
  hipMalloc((void **)&mGPU->rxn_double,
             md->n_rxn_float_param * sizeof(double));
  hipMalloc((void **)&mGPU->rxn_env_idx, (md->n_rxn + 1) * sizeof(int));
  hipMalloc((void **)&mGPU->rxn_int_indices, (md->n_rxn + 1) * sizeof(int));
  hipMalloc((void **)&mGPU->rxn_float_indices, (md->n_rxn + 1) * sizeof(int));
  hipMemcpyAsync(mGPU->rxn_int, md->rxn_int_data,
                  (md->n_rxn_int_param + md->n_rxn) * sizeof(int),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(mGPU->rxn_double, md->rxn_float_data,
                  md->n_rxn_float_param * sizeof(double),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(mGPU->rxn_env_idx, md->rxn_env_idx,
                  (md->n_rxn + 1) * sizeof(int), hipMemcpyHostToDevice,
                  stream);
  hipMemcpyAsync(mGPU->rxn_int_indices, md->rxn_int_indices,
                  (md->n_rxn + 1) * sizeof(int), hipMemcpyHostToDevice,
                  stream);
  hipMemcpyAsync(mGPU->rxn_float_indices, md->rxn_float_indices,
                  (md->n_rxn + 1) * sizeof(int), hipMemcpyHostToDevice,
                  stream);
  // Translate from int64 (sunindextype) to int
  CVDlsMem cvdls_mem = (CVDlsMem)cv_mem->cv_lmem;
  SUNMatrix J = cvdls_mem->A;
  int *jA = (int *)malloc(sizeof(int) * md->n_per_cell_solver_jac_elem);
  int *iA = (int *)malloc(sizeof(int) * (n_dep_var + 1));
  for (int i = 0; i < md->n_per_cell_solver_jac_elem; i++)
    jA[i] = SM_INDEXVALS_S(J)[i];
  for (int i = 0; i <= n_dep_var; i++) iA[i] = SM_INDEXPTRS_S(J)[i];
  hipMalloc((void **)&mGPU->djA, md->n_per_cell_solver_jac_elem * sizeof(int));
  hipMalloc((void **)&mGPU->diA, (n_dep_var + 1) * sizeof(int));
  hipMemcpyAsync(mGPU->djA, jA, md->n_per_cell_solver_jac_elem * sizeof(int),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(mGPU->diA, iA, (n_dep_var + 1) * sizeof(int),
                  hipMemcpyHostToDevice, stream);

  // Variables for each cell (struct ModelDataVariable), extracted from the
  // CVODE library
  mCPU->mdvCPU.cv_saved_tq5 = 0.;
  mCPU->mdvCPU.cv_acnrm = 0.;
  mCPU->mdvCPU.cv_eta = 1.;
  mCPU->mdvCPU.cv_hmin = 0;

  hipMalloc((void **)&mGPU->sCells, sizeof(ModelDataVariable));
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  // Metrics for statistics
  hipDeviceGetAttribute(&mGPU->clock_khz, hipDeviceAttributeClockRate, 0);
  mCPU->mdvCPU.countercvStep = 0;
  mCPU->mdvCPU.counterBCGInternal = 0;
  mCPU->mdvCPU.counterBCG = 0;
  mCPU->mdvCPU.timeNewtonIteration = 0.;
  mCPU->mdvCPU.timeJac = 0.;
  mCPU->mdvCPU.timelinsolsetup = 0.;
  mCPU->mdvCPU.timecalc_Jac = 0.;
  mCPU->mdvCPU.timef = 0.;
  mCPU->mdvCPU.timeguess_helper = 0.;
  mCPU->mdvCPU.timeBCG = 0.;
  mCPU->mdvCPU.timeDeviceCVode = 0.;
  hipMalloc((void **)&mGPU->mdvo, sizeof(ModelDataVariable));
  hipMemcpyAsync(mGPU->mdvo, &mCPU->mdvCPU, sizeof(ModelDataVariable),
                  hipMemcpyHostToDevice, stream);
#endif
  hipMemcpyAsync(&mGPU->sCells, &mCPU->mdvCPU, sizeof(ModelDataVariable),
                  hipMemcpyHostToDevice, stream);
  // Parameters for the ODE solver, extracted from CVODE library
  mGPU->cv_reltol = cv_mem->cv_reltol;
  hipMalloc((void **)&mGPU->cv_Vabstol, n_dep_var * sizeof(double));
  hipMalloc((void **)&mGPU->cv_l, L_MAX * n_cells * sizeof(double));
  hipMalloc((void **)&mGPU->cv_tau, (L_MAX + 1) * n_cells * sizeof(double));
  hipMalloc((void **)&mGPU->cv_tq, (NUM_TESTS + 1) * n_cells * sizeof(double));
  hipMalloc((void **)&mGPU->cv_last_yn, nrows * sizeof(double));
  hipMalloc((void **)&mGPU->cv_acor, nrows * sizeof(double));
  hipMalloc((void **)&mGPU->cv_acor_init, nrows * sizeof(double));
  hipMalloc((void **)&mGPU->yout, nrows * sizeof(double));
  hipMalloc((void **)&mGPU->dcv_y, nrows * sizeof(double));
  hipMalloc((void **)&mGPU->dtempv, nrows * sizeof(double));
  hipMalloc((void **)&mGPU->dtempv1, nrows * sizeof(double));
  hipMalloc((void **)&mGPU->dftemp, deriv_size);
  hipMalloc((void **)&mGPU->dewt, nrows * sizeof(double));
  hipMalloc((void **)&mGPU->dsavedJ, nnz * sizeof(double));
  double **dzn = (double **)malloc((BDF_Q_MAX + 1) * sizeof(double *));
  for (int i = 0; i <= BDF_Q_MAX; i++)
    hipMalloc(&dzn[i], nrows * sizeof(double));
  hipMalloc(&mGPU->dzn, (BDF_Q_MAX + 1) * sizeof(double *));

  hipMemcpyAsync(mGPU->cv_Vabstol, N_VGetArrayPointer(cv_mem->cv_Vabstol),
                  n_dep_var * sizeof(double), hipMemcpyHostToDevice, stream);
  for (int i = 0; i < n_cells; i++) {
    hipMemcpyAsync(mGPU->cv_l + i * L_MAX, cv_mem->cv_l,
                    L_MAX * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(mGPU->cv_tau + i * (L_MAX + 1), cv_mem->cv_tau,
                    (L_MAX + 1) * sizeof(double), hipMemcpyHostToDevice,
                    stream);
    hipMemcpyAsync(mGPU->cv_tq + i * (NUM_TESTS + 1), cv_mem->cv_tq,
                    (NUM_TESTS + 1) * sizeof(double), hipMemcpyHostToDevice,
                    stream);
  }
  for (int i = 2; i <= BDF_Q_MAX; i++)
    hipMemsetAsync(dzn[i], 0, nrows * sizeof(double), stream);
  hipMemcpy(
      mGPU->dzn, dzn, (BDF_Q_MAX + 1) * sizeof(double *),
      hipMemcpyHostToDevice);  // Synchronous because hipFree is synchronous

  for (int i = 0; i <= BDF_Q_MAX; i++) {
    hipFree(&dzn[i]);
  }
  free(dzn);

  // Parameters for the BCG solver
  hipMalloc((void **)&mGPU->dx, nrows * sizeof(double));
  hipMalloc((void **)&mGPU->ddiag, nrows * sizeof(double));
  hipMalloc((void **)&mGPU->dr0, nrows * sizeof(double));
  hipMalloc((void **)&mGPU->dr0h, nrows * sizeof(double));
  hipMalloc((void **)&mGPU->dn0, nrows * sizeof(double));
  hipMalloc((void **)&mGPU->dp0, nrows * sizeof(double));
  hipMalloc((void **)&mGPU->dt, nrows * sizeof(double));
  hipMalloc((void **)&mGPU->ds, nrows * sizeof(double));
  hipMalloc((void **)&mGPU->dy, nrows * sizeof(double));

  // Swap Jacobian format from CSC in the CPU to CSR for the GPU
  int n_row = nrows / n_cells;
  int *Ap = iA;
  int *Aj = jA;
  double *Ax = ((double *)SM_DATA_S(J));
  nnz = nnz / n_cells;
  int *Bp = (int *)malloc((n_row + 1) * sizeof(int));
  int *Bi = (int *)malloc(nnz * sizeof(int));
  double *Bx = (double *)malloc(nnz * sizeof(double));
  memset(Bp, 0, (n_row + 1) * sizeof(int));
  for (int n = 0; n < nnz; n++) {
    Bp[Aj[n]]++;
  }
  for (int col = 0, cumsum = 0; col < n_row; col++) {
    int temp = Bp[col];
    Bp[col] = cumsum;
    cumsum += temp;
  }
  Bp[n_row] = nnz;
  int *mapJSPMV = (int *)malloc(nnz * sizeof(int));
  for (int row = 0; row < n_row; row++) {
    for (int jj = Ap[row]; jj < Ap[row + 1]; jj++) {
      int col = Aj[jj];
      int dest = Bp[col];
      Bi[dest] = row;
      Bx[dest] = Ax[jj];
      mapJSPMV[jj] = dest;
      Bp[col]++;
    }
  }
  for (int col = 0, last = 0; col <= n_row; col++) {
    int temp = Bp[col];
    Bp[col] = last;
    last = temp;
  }
  nnz = md->n_per_cell_solver_jac_elem;
  int *aux_solver_id = (int *)malloc(nnz * sizeof(int));
  for (int i = 0; i < nnz; i++) {
    aux_solver_id[i] = mapJSPMV[md->jac_map[i].solver_id];
  }
  free(mapJSPMV);
  int *jac_solver_id = (int *)malloc(nnz * sizeof(int));
  JacMap *jac_map = (JacMap *)malloc(nnz * sizeof(JacMap));
  for (int i = 0; i < nnz; i++) {
    jac_solver_id[i] = aux_solver_id[i];
    aux_solver_id[i] = md->jac_map[i].solver_id;
    jac_map[i].solver_id = jac_solver_id[i];
    jac_map[i].rxn_id = md->jac_map[i].rxn_id;
    jac_map[i].param_id = md->jac_map[i].param_id;
  }
  hipMemcpyAsync(mGPU->diA, Bp, (n_row + 1) * sizeof(int),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(mGPU->djA, Bi, nnz * sizeof(int), hipMemcpyHostToDevice,
                  stream);
  hipMemcpyAsync(mGPU->jac_map, jac_map, nnz * sizeof(JacMap),
                  hipMemcpyHostToDevice, stream);
  free(Bp);
  free(Bi);
  free(Bx);
  free(jac_solver_id);
  free(aux_solver_id);
  free(jac_map);
#ifdef DEBUG_SOLVER_FAILURES
  hipMalloc((void **)&mGPU->flags, n_cells);
  malloc(mCPU->flags, n_cells);
  int *aux_solver_id = (int *)malloc(nnz * sizeof(int));
#endif
}

void free_gpu_cu(SolverData *sd) {
  ModelDataGPU *mGPU = sd->mGPU;
  mGPU = sd->mGPU;
  hipFree(mGPU->map_state_deriv);
  hipFree(mGPU->J_solver);
  hipFree(mGPU->J_state);
  hipFree(mGPU->J_deriv);
  hipFree(mGPU->rxn_int);
  hipFree(mGPU->rxn_double);
  hipFree(mGPU->state);
  hipFree(mGPU->rxn_env_data);
  hipFree(mGPU->rxn_env_idx);
  hipFree(mGPU->production_rates);
  hipFree(mGPU->loss_rates);
  hipFree(mGPU->rxn_int_indices);
  hipFree(mGPU->rxn_float_indices);
  hipFree(mGPU->jac_map);
  hipFree(mGPU->yout);
  hipFree(mGPU->cv_Vabstol);
  hipFree(mGPU->cv_l);
  hipFree(mGPU->cv_tau);
  hipFree(mGPU->cv_tq);
  hipFree(mGPU->cv_last_yn);
  hipFree(mGPU->cv_acor_init);
  hipFree(mGPU->dA);
  hipFree(mGPU->djA);
  hipFree(mGPU->diA);
  hipFree(mGPU->dx);
  hipFree(mGPU->dtempv);
  hipFree(mGPU->ddiag);
  hipFree(mGPU->dr0);
  hipFree(mGPU->dr0h);
  hipFree(mGPU->dn0);
  hipFree(mGPU->dp0);
  hipFree(mGPU->dt);
  hipFree(mGPU->ds);
  hipFree(mGPU->dy);
  hipFree(mGPU->dftemp);
  hipFree(mGPU->dcv_y);
  hipFree(mGPU->dtempv1);
  hipFree(mGPU->cv_acor);
  hipFree(mGPU->dzn);
  hipFree(mGPU->dewt);
  hipFree(mGPU->dsavedJ);
#ifdef DEBUG_SOLVER_FAILURES
  hipFree(mGPU->flags);
#endif
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  hipFree(mGPU->mdvo);
#endif
}