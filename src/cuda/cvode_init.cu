/* Copyright (C) 2021 Barcelona Supercomputing Center and University of
 * Illinois at Urbana-Champaign
 * SPDX-License-Identifier: MIT
 */

#include "cvode_cuda.h"
extern "C" {
#include "cvode_gpu.h"
}
#ifdef CAMP_USE_MPI
#include <mpi.h>
#endif

void constructor_cvode_gpu(SolverData *sd){
  ModelData *md = &(sd->model_data);
  int n_dep_var = md->n_per_cell_dep_var;
  if(n_dep_var<32) {
    printf("CAMP ERROR: TOO FEW SPECIES FOR GPU (Species < 32),"
           " use CPU case instead\n");
    exit(0);
  }
  CVodeMem cv_mem = (CVodeMem) sd->cvode_mem;
  ModelDataCPU *mCPU = &(sd->mCPU);
  CVDlsMem cvdls_mem = (CVDlsMem) cv_mem->cv_lmem;
  sd->mGPU = (ModelDataGPU *)malloc(sizeof(ModelDataGPU));
  ModelDataGPU *mGPU = sd->mGPU;
  float rate_cells_gpu = sd->gpu_percentage/100.;
  md->n_cells_gpu = md->n_cells * rate_cells_gpu;
  int n_cells = md->n_cells_gpu;
  int nrows = n_dep_var * n_cells;
  int n_state_var = md->n_per_cell_state_var;
  mGPU->n_per_cell_state_var = md->n_per_cell_state_var;
  int nGPUs;
  HANDLE_ERROR(hipGetDeviceCount(&nGPUs));
  int rank, size;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  if(rank==0){
    printf("Cells to GPU: %.d %\n",sd->gpu_percentage);
    printf("n_cells_gpu: %d\n", md->n_cells_gpu);
    printf("n_cells_cpu: %d\n", md->n_cells-md->n_cells_gpu);
  }
  int iDevice = rank % nGPUs;
  hipSetDevice(iDevice);
  mGPU->n_rxn=md->n_rxn;
  mGPU->n_rxn_env_data=md->n_rxn_env_data;
  mGPU->cv_reltol = cv_mem->cv_reltol;
  HANDLE_ERROR(hipMalloc((void **) &mGPU->state, n_state_var * n_cells * sizeof(double)));
  hipMalloc((void **) &mGPU->rxn_env_data, md->n_rxn_env_data * n_cells * sizeof(double));
  int num_spec = n_dep_var*n_cells;
  hipMalloc((void **) &(mGPU->production_rates),num_spec*sizeof(mGPU->production_rates));
  hipMalloc((void **) &(mGPU->loss_rates),num_spec*sizeof(mGPU->loss_rates));
  hipMalloc((void **) &mGPU->map_state_deriv, n_dep_var * sizeof(int));
  int *map_state_derivCPU = (int *)malloc(n_dep_var * sizeof(int));
  int i_dep_var = 0;
  for (int i_spec = 0; i_spec < n_state_var; i_spec++) {
    if (md->var_type[i_spec] == CHEM_SPEC_VARIABLE) {
      map_state_derivCPU[i_dep_var] = i_spec;
      i_dep_var++;
    }
  }
  hipMemcpy(mGPU->map_state_deriv, map_state_derivCPU,
             n_dep_var * sizeof(int), hipMemcpyHostToDevice);
  free(map_state_derivCPU);
  size_t deriv_size = n_dep_var * n_cells * sizeof(double);
  int nnz = md->n_per_cell_solver_jac_elem * n_cells;
  size_t jac_size = nnz * sizeof(double);
  hipMalloc((void **) &mGPU->dA, jac_size);
  hipMalloc((void **) &mGPU->J_solver, jac_size);
  hipMalloc((void **) &mGPU->J_state, deriv_size);
  double *J_state = N_VGetArrayPointer(md->J_state);
  hipMemset(mGPU->J_state, 0, deriv_size);
  hipMalloc((void **) &mGPU->J_deriv, deriv_size);
  double *J_deriv = N_VGetArrayPointer(md->J_deriv);
  hipMemset(mGPU->J_deriv, 0, deriv_size);
  hipMemset(mGPU->J_solver, 0, jac_size);
  hipMalloc((void **) &mGPU->jac_map, sizeof(JacMap) * md->n_mapped_values);
  hipMalloc((void **) &mGPU->n_mapped_values, 1 * sizeof(int));
  hipMemcpy(mGPU->jac_map, md->jac_map, sizeof(JacMap) * md->n_mapped_values, hipMemcpyHostToDevice);
  hipMemcpy(mGPU->n_mapped_values, &md->n_mapped_values, 1 * sizeof(int), hipMemcpyHostToDevice);
  Jacobian *jac = &sd->jac;
  JacobianGPU *jacgpu = &(mGPU->jac);
  hipMalloc((void **) &jacgpu->num_elem, 1 * sizeof(jacgpu->num_elem));
  hipMemcpy(jacgpu->num_elem, &jac->num_elem, 1 * sizeof(jacgpu->num_elem), hipMemcpyHostToDevice);
  int num_elem = jac->num_elem * n_cells;
  hipMalloc((void **) &(jacgpu->production_partials), num_elem * sizeof(double));
  hipMalloc((void **) &(jacgpu->loss_partials), num_elem * sizeof(double));
  hipMemset(jacgpu->production_partials, 0, num_elem * sizeof(double));
  hipMemset(jacgpu->loss_partials, 0, num_elem * sizeof(double));
  hipMalloc((void **) &mGPU->rxn_int, (md->n_rxn_int_param + md->n_rxn)*sizeof(int));
  hipMalloc((void **) &mGPU->rxn_double, md->n_rxn_float_param*sizeof(double));
  hipMalloc((void **) &mGPU->rxn_env_idx, (md->n_rxn+1) * sizeof(int));
  hipMalloc((void **) &mGPU->rxn_int_indices, (md->n_rxn+1)*sizeof(int));
  hipMalloc((void **) &mGPU->rxn_float_indices, (md->n_rxn+1)*sizeof(int));
  hipMemcpy(mGPU->rxn_int, md->rxn_int_data,(md->n_rxn_int_param + md->n_rxn)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->rxn_double, md->rxn_float_data, md->n_rxn_float_param*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->rxn_env_idx, md->rxn_env_idx, (md->n_rxn+1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->rxn_int_indices, md->rxn_int_indices,(md->n_rxn+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->rxn_float_indices, md->rxn_float_indices,(md->n_rxn+1)*sizeof(int), hipMemcpyHostToDevice);
  double ** dr0 = &mGPU->dr0;
  double ** dr0h = &mGPU->dr0h;
  double ** dn0 = &mGPU->dn0;
  double ** dp0 = &mGPU->dp0;
  double ** dt = &mGPU->dt;
  double ** ds = &mGPU->ds;
  double ** dy = &mGPU->dy;
  double ** ddiag = &mGPU->ddiag;
  hipMalloc(dr0,nrows*sizeof(double));
  hipMalloc(dr0h,nrows*sizeof(double));
  hipMalloc(dn0,nrows*sizeof(double));
  hipMalloc(dp0,nrows*sizeof(double));
  hipMalloc(dt,nrows*sizeof(double));
  hipMalloc(ds,nrows*sizeof(double));
  hipMalloc(dy,nrows*sizeof(double));
  hipMalloc(ddiag,nrows*sizeof(double));
  hipMalloc((void **) &mGPU->dsavedJ, nnz * sizeof(double));
  //Translate from int64 (sunindextype) to int
  SUNMatrix J = cvdls_mem->A;
  int *jA = (int *) malloc(sizeof(int) * md->n_per_cell_solver_jac_elem);
  int *iA = (int *) malloc(sizeof(int) * (n_dep_var + 1));
  for (int i = 0; i < md->n_per_cell_solver_jac_elem; i++)
    jA[i] = SM_INDEXVALS_S(J)[i];
  for (int i = 0; i <= n_dep_var; i++)
    iA[i] = SM_INDEXPTRS_S(J)[i];
  hipMalloc((void **) &mGPU->djA, md->n_per_cell_solver_jac_elem * sizeof(int));
  hipMalloc((void **) &mGPU->diA, (n_dep_var + 1) * sizeof(int));
  hipMemcpy(mGPU->djA, jA, md->n_per_cell_solver_jac_elem * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->diA, iA, (n_dep_var + 1) * sizeof(int), hipMemcpyHostToDevice);
  hipMalloc((void **) &mGPU->dftemp, deriv_size);
  hipMalloc((void **) &mGPU->sCells, sizeof(ModelDataVariable));
  hipMalloc((void **) &mGPU->dewt, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->cv_acor, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->dtempv, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->dtempv1, nrows * sizeof(double));
  sd->dzn=(double**)malloc((BDF_Q_MAX + 1)*sizeof(double*));
  for(int i=0;i<=BDF_Q_MAX;i++){
    hipMalloc(&sd->dzn[i], nrows * sizeof(double));
  }
  hipMalloc(&mGPU->dzn, (BDF_Q_MAX + 1) * sizeof(double*));
  hipMemcpy(mGPU->dzn, sd->dzn, (BDF_Q_MAX + 1) * sizeof(double*), hipMemcpyHostToDevice);
  for (int i = 2; i <= BDF_Q_MAX; i++) {
    hipMemset(sd->dzn[i], 0, nrows * sizeof(double));
  }
  hipMalloc((void **) &mGPU->dcv_y, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->dx, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->cv_last_yn, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->cv_acor_init, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->cv_acor, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->yout, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->cv_l, L_MAX * n_cells * sizeof(double));
  hipMalloc((void **) &mGPU->cv_tau, (L_MAX + 1) * n_cells * sizeof(double));
  hipMalloc((void **) &mGPU->cv_tq, (NUM_TESTS + 1) * n_cells * sizeof(double));
  mCPU->mdvCPU.cv_saved_tq5 = 0.;
  mCPU->mdvCPU.cv_acnrm = 0.;
  mCPU->mdvCPU.cv_eta = 1.;
  mCPU->mdvCPU.cv_hmin = 0;
  hipMemcpy(&mGPU->sCells, &mCPU->mdvCPU, sizeof(ModelDataVariable), hipMemcpyHostToDevice);
  for (int i = 0; i < n_cells; i++) {
    hipMemcpy(mGPU->cv_l + i * L_MAX, cv_mem->cv_l, L_MAX * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(mGPU->cv_tau + i * (L_MAX + 1), cv_mem->cv_tau, (L_MAX + 1) * sizeof(double),
                    hipMemcpyHostToDevice);
    hipMemcpy(mGPU->cv_tq + i * (NUM_TESTS + 1), cv_mem->cv_tq, (NUM_TESTS + 1) * sizeof(double),
                    hipMemcpyHostToDevice);
  }
  hipMalloc((void **) &mGPU->cv_Vabstol, n_dep_var * sizeof(double));
  double *cv_Vabstol = N_VGetArrayPointer(cv_mem->cv_Vabstol);
  hipMemcpy(mGPU->cv_Vabstol, cv_Vabstol, n_dep_var * sizeof(double), hipMemcpyHostToDevice);
#ifdef PROFILE_SOLVING
  hipEventCreate(&sd->startGPU);
  hipEventCreate(&sd->stopGPU);
  sd->timeSync=0;
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  hipMalloc((void **) &mGPU->mdvo, sizeof(ModelDataVariable));
  hipDeviceGetAttribute(&mGPU->clock_khz, hipDeviceAttributeClockRate, 0);
  mCPU->mdvCPU.countercvStep=0;
  mCPU->mdvCPU.counterBCGInternal=0;
  mCPU->mdvCPU.counterBCG=0;
  mCPU->mdvCPU.timeNewtonIteration=0.;
  mCPU->mdvCPU.timeJac=0.;
  mCPU->mdvCPU.timelinsolsetup=0.;
  mCPU->mdvCPU.timecalc_Jac=0.;
  mCPU->mdvCPU.timef=0.;
  mCPU->mdvCPU.timeguess_helper=0.;
  mCPU->mdvCPU.dtBCG=0.;
  mCPU->mdvCPU.dtcudaDeviceCVode=0.;
  mCPU->mdvCPU.dtPostBCG=0.;
  hipMemcpy(mGPU->mdvo, &mCPU->mdvCPU, sizeof(ModelDataVariable), hipMemcpyHostToDevice);
#endif
#endif
//Swap CSC to CSR
  int n_row=nrows/n_cells;
  int* Ap=iA;
  int* Aj=jA;
  double* Ax=((double *) SM_DATA_S(J));
  nnz=nnz/n_cells;
  int* Bp=(int*)malloc((n_row+1)*sizeof(int));
  int* Bi=(int*)malloc(nnz*sizeof(int));
  double* Bx=(double*)malloc(nnz*sizeof(double));
  memset(Bp, 0, (n_row+1)*sizeof(int));
  for (int n = 0; n < nnz; n++){
   Bp[Aj[n]]++;
  }
  for(int col = 0, cumsum = 0; col < n_row; col++){
    int temp  = Bp[col];
    Bp[col] = cumsum;
    cumsum += temp;
  }
  Bp[n_row] = nnz;
  int *mapJSPMV= (int *)malloc(nnz * sizeof(int));
  for(int row = 0; row < n_row; row++){
    for(int jj = Ap[row]; jj < Ap[row+1]; jj++){
      int col  = Aj[jj];
      int dest = Bp[col];
      Bi[dest] = row;
      Bx[dest] = Ax[jj];
      mapJSPMV[jj]=dest;
      Bp[col]++;
    }
  }
  for(int col = 0, last = 0; col <= n_row; col++){
    int temp  = Bp[col];
    Bp[col] = last;
    last    = temp;
  }
  nnz=md->n_mapped_values;
  int *aux_solver_id= (int *)malloc(nnz * sizeof(int));
  for (int i = 0; i < nnz; i++){
    aux_solver_id[i]=mapJSPMV[md->jac_map[i].solver_id];
  }
  free(mapJSPMV);
  int *jac_solver_id= (int *)malloc(nnz * sizeof(int));
  JacMap *jac_map = (JacMap *)malloc(nnz*sizeof(JacMap));
  for (int i = 0; i < nnz; i++){
    jac_solver_id[i]=aux_solver_id[i];
    aux_solver_id[i]=md->jac_map[i].solver_id;
    jac_map[i].solver_id=jac_solver_id[i];
    jac_map[i].rxn_id=md->jac_map[i].rxn_id;
    jac_map[i].param_id=md->jac_map[i].param_id;

  }
  hipMemcpy(mGPU->diA, Bp, (n_row + 1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->djA, Bi, nnz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->jac_map, jac_map, nnz * sizeof(JacMap), hipMemcpyHostToDevice);
  free(Bp);
  free(Bi);
  free(Bx);
  free(jac_solver_id);
  free(aux_solver_id);
  free(jac_map);
}

void free_gpu_cu(SolverData *sd) {
  ModelDataGPU *mGPU = sd->mGPU;
  mGPU = sd->mGPU;
  hipFree(mGPU->map_state_deriv);
  hipFree(mGPU->J_solver);
  hipFree(mGPU->J_state);
  hipFree(mGPU->J_deriv);
  hipFree(mGPU->rxn_int);
  hipFree(mGPU->rxn_double);
  hipFree(mGPU->state);
  hipFree(mGPU->rxn_env_data);
  hipFree(mGPU->rxn_env_idx);
  hipFree(mGPU->production_rates);
  hipFree(mGPU->loss_rates);
  hipFree(mGPU->rxn_int_indices);
  hipFree(mGPU->rxn_float_indices);
  hipFree(mGPU->n_mapped_values);
  hipFree(mGPU->jac_map);
  hipFree(mGPU->yout);
  hipFree(mGPU->cv_Vabstol);
  hipFree(mGPU->cv_l);
  hipFree(mGPU->cv_tau);
  hipFree(mGPU->cv_tq);
  hipFree(mGPU->cv_last_yn);
  hipFree(mGPU->cv_acor_init);
  hipFree(mGPU->dA);
  hipFree(mGPU->djA);
  hipFree(mGPU->diA);
  hipFree(mGPU->dx);
  hipFree(mGPU->dtempv);
  hipFree(mGPU->ddiag);
  hipFree(mGPU->dr0);
  hipFree(mGPU->dr0h);
  hipFree(mGPU->dn0);
  hipFree(mGPU->dp0);
  hipFree(mGPU->dt);
  hipFree(mGPU->ds);
  hipFree(mGPU->dy);
  hipFree(mGPU->dftemp);
  hipFree(mGPU->dcv_y);
  hipFree(mGPU->dtempv1);
  hipFree(mGPU->cv_acor);
  for(int i=0;i<=BDF_Q_MAX;i++){
    hipFree(&sd->dzn[i]);
  }
  hipFree(mGPU->dzn);
  free(sd->dzn);
  hipFree(mGPU->dewt);
  hipFree(mGPU->dsavedJ);
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  hipFree(mGPU->mdvo);
#endif
}